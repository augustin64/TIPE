#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#include <omp.h>

#include "../src/cnn/include/matrix_multiplication.h"
#include "../src/common/include/memory_management.h"
#include "../src/common/include/colors.h"
#include "../src/common/include/utils.h"


float random_float(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float **matrix, int n, int p) {
  for (int i=0; i < n; i++) {
    for (int j=0; j < p; j++) {
        matrix[i][j] = random_float(0.0f, 15.0f);
    }
  }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float** create_matrix(int n, int p) {
    float** matrix = (float**)nalloc(n, sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)nalloc(p, sizeof(float));
    }

    fill_matrix_random(matrix, n, p);
    return matrix;
}


float** create_empty_matrix(int n, int p) {
    float** matrix = (float**)nalloc(n, sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)nalloc(p, sizeof(float));
        for (int j=0; j < p; j++) {
            matrix[i][j] = 0.;
        }
    }
    return matrix;
}


bool check_matrices_equality(float** m1, float** m2, int n, int p, int acceptation) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            if (fabs(m1[i][j] - m2[i][j]) > 0.01*acceptation) {
                return false;
            }
        }
    }
    return true;
}

void run_matrices_test(int n, int p, int q) {
    double start_time, end_time;
    double cpu_time_used, gpu_time_used;

    float** matrix1 = create_matrix(n, p);
    float** matrix2 = create_matrix(p, q);
    float** result_gpu = create_empty_matrix(n, q);
    float** result_cpu = create_empty_matrix(n, q);

    printf("(%d,%d)x(%d,%d) Data generation complete.\n", n, p, p, q);
    start_time = omp_get_wtime();
    matrix_multiplication_device(matrix1, matrix2, result_gpu, n, p, q);
    end_time = omp_get_wtime();

    cpu_time_used = end_time - start_time;
    printf("(%d,%d)x(%d,%d) Time used for GPU: %lf seconds\n", n, p, p, q, cpu_time_used);
    
    start_time = omp_get_wtime();
    matrix_multiplication_host(matrix1, matrix2, result_cpu, n, p, q);
    end_time = omp_get_wtime();

    gpu_time_used = end_time - start_time;
    printf("(%d,%d)x(%d,%d) Time used for CPU: %lf seconds\n", n, p, p, q, gpu_time_used);

    // Vérification de l'égalité des matrices
    printf("(%d,%d)x(%d,%d) Checking equality.\n", n, p, p, q);
    if (!check_matrices_equality(result_gpu, result_cpu, n, q, p)) {
        exit(1);
    }
    printf(GREEN "OK\n" RESET);

    // On libère l'espace mémoire alloué
    for (int i=0; i < n; i++) {
        gree(matrix1[i], false);
    }
    gree(matrix1, false);

    for (int i=0; i < p; i++) {
        gree(matrix2[i], false);
    }
    gree(matrix2, false);

    for (int i=0; i < n; i++) {
        gree(result_cpu[i], false);
    }
    gree(result_cpu, false);

    for (int i=0; i < n; i++) {
        gree(result_gpu[i], false);
    }
    gree(result_gpu, false);
}


int main() {
    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = cuda_setup(true);
    if (!cuda_compatible) {
        printf(RED "CUDA not compatible, skipping tests.\n" RESET);
        return 0;
    }
    printf(GREEN "OK\n" RESET);

    srand(time(NULL));
    run_matrices_test(200, 1000, 200);
    run_matrices_test(200, 1000, 20);
    run_matrices_test(20, 1000, 200);
    
    return 0;
}

// On obtient une différence entre le calcul fait par le GPU et par le CPU.
// Cette différence est linéaire en p. (err_moy = p*1.639e-6)
// Elle ne varie pas en fonction de n et q.
// Cette erreur est sûrement dûe à différences mineurs dans la précision du stockage des flottants
// dans la mémoire RAM et VRAM (du GPU)