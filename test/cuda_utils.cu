#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#ifdef __HIPCC__
#   warning compiling for CUDA
#   include "../src/mnist/cuda_utils.cu"
#   define MAX_CUDA_THREADS 1024 // from NVIDIA documentation
#else
#   warning skipping CUDA checks
#endif

#ifdef __HIPCC__
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void check_labels(int n, unsigned int* labels) {
    for (int i=0; i < n; i++) {
        (void)labels[i];
    }
}


int main() {
    printf("Test de la compatibilité CUDA\n");
    check_cuda_compatibility();
    printf("OK\n");

    printf("Lecture des labels\n");
    unsigned int* labels = cudaReadMnistLabels("data/mnist/t10k-labels-idx1-ubyte");
    printf("OK\n");

    printf("Test des labels\n");
    //! TODO: fix
    // Ne provoque pas d'erreurs, mais tous les labels valent 1
    check_labels<<<1, 1>>>(10000, labels);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    printf("OK\n");
    
    return 0;
}
#else
int main() {
    printf("Pas de test CUDA à passer\n");
    return 0;
}
#endif