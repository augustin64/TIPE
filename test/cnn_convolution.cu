#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <assert.h>
#include <math.h>
#include <time.h>
#include <omp.h>

#include "../src/common/include/memory_management.h"
#include "../src/cnn/include/convolution.h"
#include "../src/common/include/colors.h"
#include "../src/common/include/utils.h"
#include "../src/cnn/include/struct.h"


float random_float(float low, float high) {
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float ***matrix, int n, int p, int q, float max_val) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            for (int k=0; k < q; k++) {
                matrix[i][j][k] = random_float(0.0f, max_val);
            }
        }
    }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float*** create_matrix(int n, int p, int q, float max_val) {
    float*** matrix = (float***)nalloc(n, sizeof(float**));
    for (int i=0; i < n; i++) {
        matrix[i] = (float**)nalloc(p, sizeof(float*));
        for (int j=0; j < p; j++) {
            matrix[i][j] = (float*)nalloc(q, sizeof(float));
        }
    }

    fill_matrix_random(matrix, n, p, q, max_val);
    return matrix;
}


float*** create_empty_matrix(int n, int p, int q) {
    float*** matrix = (float***)nalloc(n, sizeof(float**));
    for (int i=0; i < n; i++) {
        matrix[i] = (float**)nalloc(p, sizeof(float*));
        for (int j=0; j < p; j++) {
            matrix[i][j] = (float*)nalloc(q, sizeof(float));
            for (int k=0; k < q; k++) {
                matrix[i][j][k] = 0.;
            }
        }
    }
    return matrix;
}

void free_matrix(float*** matrix, int n, int p) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            gree(matrix[i][j], false);
        }
        gree(matrix[i], false);
    }
    gree(matrix, false);
}

bool check_matrices_equality(float*** m1, float*** m2, int n, int p, int q, int acceptation) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            for (int k=0; k < q; k++) {
                if (fabs(m1[i][j][k] - m2[i][j][k]) > 0.01*acceptation) {
                    printf(RED "diff %d %d %d: %f val: %f et %f\n" RESET, i, j, k, fabs(m1[i][j][k] - m2[i][j][k]), m1[i][j][k], m2[i][j][k]);
                    return false;
                }
            }
        }
    }
    return true;
}

void run_convolution_test(int input_width, int output_width, int rows, int columns) {
    assert(input_width >= output_width);
    int k_size = input_width - output_width +1;

    // Génération des données aléatoires
    Kernel_cnn* kernel = (Kernel_cnn*)nalloc(1, sizeof(Kernel_cnn));
    
    kernel->k_size = k_size;
    kernel->rows = rows;
    kernel->columns = columns;

    // bias[kernel->columns][output_width][output_width]
    kernel->bias = create_matrix(kernel->columns, output_width, output_width, 15.0f);
    kernel->d_bias = create_matrix(kernel->columns, output_width, output_width, 1.5f);
    #ifdef ADAM_CNN_BIAS
    kernel->s_d_bias = create_matrix(kernel->columns, output_width, output_width, 1.5f);
    kernel->v_d_bias = create_matrix(kernel->columns, output_width, output_width, 1.5f);
    #endif

    // weights[rows][columns][k_size][k_size]
    kernel->weights = (float****)nalloc(kernel->rows, sizeof(float***));
    kernel->d_weights = (float****)nalloc(kernel->rows, sizeof(float***));
    #ifdef ADAM_CNN_WEIGHTS
    kernel->s_d_weights = (float****)nalloc(kernel->rows, sizeof(float***));
    kernel->v_d_weights = (float****)nalloc(kernel->rows, sizeof(float***));
    #endif
    for (int i=0; i < kernel->rows; i++) {
        kernel->weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 15.0f);
        kernel->d_weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 1.5f);
        #ifdef ADAM_CNN_WEIGHTS
        kernel->s_d_weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 1.5f);
        kernel->v_d_weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 1.5f);
        #endif
    }

    float*** input = create_matrix(kernel->rows, input_width, input_width, 5.0f);
    float*** output_cpu = create_empty_matrix(kernel->columns, output_width, output_width);
    float*** output_gpu = create_empty_matrix(kernel->columns, output_width, output_width);

    printf("(%d, %d, %d, %d) Data generation complete\n", rows, columns, input_width, output_width);


    // Lancement des calculs
    double start_time, end_time;
    double cpu_time_used, gpu_time_used;

    start_time = omp_get_wtime();
    make_convolution_device(kernel, input, output_gpu, output_width, 1, 0);
    end_time = omp_get_wtime();


    gpu_time_used = end_time - start_time;
    printf("(%d, %d, %d, %d) Time used for GPU: %lf seconds\n", rows, columns, input_width, output_width, gpu_time_used);


    start_time = omp_get_wtime();
    make_convolution_cpu(kernel, input, output_cpu, output_width, 1, 0);
    end_time = omp_get_wtime();

    cpu_time_used = end_time - start_time;
    printf("(%d, %d, %d, %d) Time used for CPU: %lf seconds\n", rows, columns, input_width, output_width, cpu_time_used);    

    // Vérification de l'égalité des matrices
    printf("(%d, %d, %d, %d) Checking equality.\n", rows, columns, input_width, output_width);
    if (!check_matrices_equality(output_gpu, output_cpu, kernel->columns, output_width, output_width, kernel->k_size)) {// TODO: change acceptation
        exit(1);
    }
    printf(GREEN "OK\n" RESET);

    free_matrix(kernel->bias, kernel->columns, output_width);
    free_matrix(kernel->d_bias, kernel->columns, output_width);
    #ifdef ADAM_CNN_BIAS
    free_matrix(kernel->s_d_bias, kernel->columns, output_width);
    free_matrix(kernel->v_d_bias, kernel->columns, output_width);
    #endif

    for (int i=0; i < kernel->rows; i++) {
        free_matrix(kernel->weights[i], kernel->columns, kernel->k_size);
        free_matrix(kernel->d_weights[i], kernel->columns, kernel->k_size);
        #ifdef ADAM_CNN_WEIGHTS
        free_matrix(kernel->s_d_weights[i], kernel->columns, kernel->k_size);
        free_matrix(kernel->v_d_weights[i], kernel->columns, kernel->k_size);
        #endif
    }
    gree(kernel->weights, false);
    gree(kernel->d_weights, false);
    #ifdef ADAM_CNN_WEIGHTS
    gree(kernel->s_d_weights, false);
    gree(kernel->v_d_weights, false);
    #endif

    free_matrix(input, kernel->rows, input_width);
    free_matrix(output_cpu, kernel->columns, output_width);
    free_matrix(output_gpu, kernel->columns, output_width);
}


int main() {
    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = cuda_setup(true);
    if (!cuda_compatible) {
        printf(RED "CUDA not compatible, skipping tests.\n" RESET);
        return 0;
    }
    printf(GREEN "OK\n" RESET);
    
    srand(time(NULL));

    run_convolution_test(20, 15, 30, 40);
    run_convolution_test(30, 25, 40, 50);
    run_convolution_test(250, 200, 3, 3);
    
    return 0;
}