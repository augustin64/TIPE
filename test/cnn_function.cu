#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>

#include "../src/include/memory_management.h"
#include "../src/cnn/include/function.h"
#include "../src/include/colors.h"
#include "../src/include/utils.h"


int main() {
    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = check_cuda_compatibility();
    if (!cuda_compatible) {
        printf(RED "CUDA not compatible, skipping tests.\n" RESET);
        return 0;
    }
    printf(GREEN "OK\n" RESET);

    printf("Initialisation OK\n");
    // Initialise values
    int depth = 10;
    int rows = 10;
    int columns = 10;

    float*** input = (float***)nalloc(depth, sizeof(float**));
    float*** input_initial = (float***)malloc(depth*sizeof(float**));
    for (int i=0; i < depth; i++) {
        input[i] = (float**)nalloc(rows, sizeof(float*));
        input_initial[i] = (float**)malloc(rows*sizeof(float*));
        for (int j=0; j < rows; j++) {
            input[i][j] = (float*)nalloc(columns, sizeof(float));
            input_initial[i][j] = (float*)malloc(columns*sizeof(float));
            for (int k=0; k < columns; k++) {
                input[i][j][k] = rand()/RAND_MAX;
                input_initial[i][j][k] = input[i][j][k];
            }
        }
    }
    printf(GREEN "OK\n" RESET);

    funcPtr func = get_activation_function(TANH);

    printf("Calcul par CUDA\n");
    apply_function_input(TANH, input, depth, rows, columns);
    printf(GREEN "OK\n" RESET);

    printf("Vérification des résultats\n");
    for (int i=0; i < depth; i++) {
        for (int j=0; j < rows; j++) {
            for (int k=0; k < columns; k++) {
                if (fabs((*func)(input_initial[i][j][k]) - input[i][j][k]) > 1e-6) {
                    printf_error((char*)"Les résultats ne coincident pas\n");
                    printf("Différence %e\n", fabs((*func)(input_initial[i][j][k]) - input[i][j][k]));
                    //exit(1);
                }
            }
            gree(input[i][j]);
            free(input_initial[i][j]);
        }
        gree(input[i]);
        free(input_initial[i]);
    }
    gree(input);
    free(input_initial);

    printf(GREEN "OK\n" RESET);
    return 0;
}