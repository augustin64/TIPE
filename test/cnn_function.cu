#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>

#include "../src/include/memory_management.h"
#include "../src/cnn/include/function.h"
#include "../src/include/colors.h"
#include "../src/include/utils.h"

#include "../src/cnn/include/config.h"

__global__ void local_kernel(funcPtr f, float*** input, int depth, int rows, int columns) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < rows
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < columns

    if (idx >= depth || idy >= rows || idz >= columns) {
        return;
    }

    input[idx][idy][idz] = (*f)(input[idx][idy][idz]);
}


void test1(int activation, bool use_local_kernel) {
    printf("Test sur la fonction %d\n", activation);
    printf("\tInitialisation OK\n");
    // Initialise values
    int depth = 10;
    int rows = 10;
    int columns = 10;

    float*** input = (float***)nalloc(depth, sizeof(float**));
    float*** input_initial = (float***)malloc(depth*sizeof(float**));
    for (int i=0; i < depth; i++) {
        input[i] = (float**)nalloc(rows, sizeof(float*));
        input_initial[i] = (float**)malloc(rows*sizeof(float*));
        for (int j=0; j < rows; j++) {
            input[i][j] = (float*)nalloc(columns, sizeof(float));
            input_initial[i][j] = (float*)malloc(columns*sizeof(float));
            for (int k=0; k < columns; k++) {
                input[i][j][k] = rand()/(float)RAND_MAX;
                input_initial[i][j][k] = input[i][j][k];
            }
        }
    }
    printf("\t" GREEN "OK\n" RESET);

    funcPtr func_cpu = get_activation_function(activation);

    if (!use_local_kernel) {
        printf("\tCalcul par CUDA\n");
        apply_function_input(activation, input, depth, rows, columns);
    } else {
        printf("\tCalcul par CUDA sur le kernel local\n");
        dim3 gridSize(i_div_up(depth, BLOCKSIZE_x), i_div_up(rows, BLOCKSIZE_y), i_div_up(columns, BLOCKSIZE_z));
        dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

        funcPtr function_cuda = get_activation_function_cuda(activation);

        local_kernel<<<gridSize, blockSize>>>(function_cuda, input, depth, rows, columns);
        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
    printf("\t" GREEN "OK\n" RESET);

    printf("\tVérification des résultats\n");
    for (int i=0; i < depth; i++) {
        for (int j=0; j < rows; j++) {
            for (int k=0; k < columns; k++) {
                if (fabs((*func_cpu)(input_initial[i][j][k]) - input[i][j][k]) > 1e-6) {
                    printf_error((char*)"Les résultats ne coincident pas\n");
                    printf("Différence %e\n", fabs((*func_cpu)(input_initial[i][j][k]) - input[i][j][k]));
                    exit(1);
                }
            }
            gree(input[i][j]);
            free(input_initial[i][j]);
        }
        gree(input[i]);
        free(input_initial[i]);
    }
    gree(input);
    free(input_initial);

    printf("\t" GREEN "OK\n" RESET);
    printf(GREEN "OK\n" RESET);
}

int main() {
    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = check_cuda_compatibility();
    if (!cuda_compatible) {
        printf(RED "CUDA not compatible, skipping tests.\n" RESET);
        return 0;
    }
    printf(GREEN "OK\n" RESET);

    for (int i=1; i < 7; i++) {
        if (i != 5) { // Exclude SOFTMAX
            test1(i, false); // use function i
            test1(-i, false); // use function i'
            test1(i, true); // use function i in the kernel declared in this file
            test1(-i, true); // use function i' in the kernel declared in this file
        }
    }
    return 0;
}