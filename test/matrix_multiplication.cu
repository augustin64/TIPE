#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>

#include "../src/cnn/matrix_multiplication.cu"


float random_float(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float **matrix, int n, int p) {
  for (int i=0; i < n; i++) {
    for (int j=0; j < p; j++) {
        matrix[i][j] = random_float(0.0f, 15.0f);
    }
  }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float** create_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(sizeof(float)*p);
    }

    fill_matrix_random(matrix, n, p);
    return matrix;
}


float** create_empty_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(p*sizeof(float));
        for (int j=0; j < p; j++) {
            matrix[i][j] = 0.;
        }
    }
    return matrix;
}

float max_float(float a, float b) {
    return a > b ? a : b;
}


bool check_matrices_equality(float** m1, float** m2, int n, int p) {
    float err_max = 0.;
    float err_moy = 0.;
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            if (fabs(m1[i][j] - m2[i][j]) > 0.8) {
                //printf("%d %d\n", i, j);
                //return false;
            }
            err_max = max_float(err_max, fabs(m1[i][j] - m2[i][j]));
            err_moy += fabs(m1[i][j] - m2[i][j]);
        }
    }
    printf("err_max: %f\n", err_max);
    printf("err_moy: %f\n", err_moy/(n*p));
    return true;
}


int main() {
    clock_t start, end;
    double cpu_time_used;

    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = check_cuda_compatibility();
    if (!cuda_compatible) {
        printf("CUDA not compatible, skipping tests.\n");
        return 0;
    }
    printf("OK\n");


    printf("Generating matrices.\n");
    srand(time(NULL));
    int n = 200;
    int p = 1000;
    int q = 200;
    float** matrix1 = create_matrix(n, p);
    float** matrix2 = create_matrix(p, q);
    float** result_gpu = create_empty_matrix(n, q);
    float** result_cpu = create_empty_matrix(n, q);
    printf("OK\n");

    printf("Computing on GPU.\n");
    start = clock();
    matrix_multiplication_device(matrix1, matrix2, result_gpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time used for GPU: %lf seconds\n", cpu_time_used);
    printf("OK\n");
    
    printf("Computing on CPU.\n");
    start = clock();
    matrix_multiplication_host(matrix1, matrix2, result_cpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time used for CPU: %lf seconds\n", cpu_time_used);
    printf("OK\n");


    printf("Checking equality.\n");
    if (!check_matrices_equality(result_gpu, result_cpu, n, q)) {
        return 1;
    }
    printf("OK\n");
    
    return 0;
}

// On obtient une différence entre le calcul fait par le GPU et par le CPU.
// Cette différence est linéaire en p. (err_moy = p*1.639e-6)
// Elle ne varie pas en fonction de n et q.
// Cette erreur est sûrement dûe à différences mineurs dans la précision du stockage des flottants
// Dans la mémoire RAM et VRAM (du GPU)