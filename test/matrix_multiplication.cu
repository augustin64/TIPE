#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>

#include "../src/cnn/matrix_multiplication.cu"


float random_float(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float **matrix, int n, int p) {
  for (int i=0; i < n; i++) {
    for (int j=0; j < p; j++) {
        matrix[i][j] = random_float(0.0f, 15.0f);
    }
  }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float** create_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(sizeof(float)*p);
    }

    fill_matrix_random(matrix, n, p);
    return matrix;
}


float** create_empty_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(p*sizeof(float));
        for (int j=0; j < p; j++) {
            matrix[i][j] = 0.;
        }
    }
    return matrix;
}


bool check_matrices_equality(float** m1, float** m2, int n, int p, int acceptation) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            if (fabs(m1[i][j] - m2[i][j]) > 0.01*acceptation) {
                return false;
            }
        }
    }
    return true;
}

void run_matrices_test(int n, int p, int q) {
    clock_t start, end;
    double cpu_time_used;

    float** matrix1 = create_matrix(n, p);
    float** matrix2 = create_matrix(p, q);
    float** result_gpu = create_empty_matrix(n, q);
    float** result_cpu = create_empty_matrix(n, q);

    printf("(%d,%d)x(%d,%d) Computing on GPU.\n", n, p, p, q);
    start = clock();
    matrix_multiplication_device(matrix1, matrix2, result_gpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("(%d,%d)x(%d,%d) Time used for GPU: %lf seconds\n", n, p, p, q, cpu_time_used);
    printf("OK\n");
    
    printf("(%d,%d)x(%d,%d) Computing on CPU.\n", n, p, p, q);
    start = clock();
    matrix_multiplication_host(matrix1, matrix2, result_cpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("(%d,%d)x(%d,%d) Time used for CPU: %lf seconds\n", n, p, p, q, cpu_time_used);
    printf("OK\n");

    // Vérification de l'égalité des matrices
    printf("(%d,%d)x(%d,%d) Checking equality.\n", n, p, p, q);
    if (!check_matrices_equality(result_gpu, result_cpu, n, q, p)) {
        exit(1);
    }
    printf("OK\n");

    // On libère l'espace mémoire alloué
    for (int i=0; i < n; i++) {
        free(matrix1[i]);
    }
    free(matrix1);

    for (int i=0; i < p; i++) {
        free(matrix2[i]);
    }
    free(matrix2);

    for (int i=0; i < n; i++) {
        free(result_cpu[i]);
    }
    free(result_cpu);

    for (int i=0; i < n; i++) {
        free(result_gpu[i]);
    }
    free(result_gpu);
}


int main() {
    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = check_cuda_compatibility();
    if (!cuda_compatible) {
        printf("CUDA not compatible, skipping tests.\n");
        return 0;
    }
    printf("OK\n");

    srand(time(NULL));
    run_matrices_test(200, 1000, 200);
    run_matrices_test(200, 1000, 20);
    run_matrices_test(20, 1000, 200);
    
    return 0;
}

// On obtient une différence entre le calcul fait par le GPU et par le CPU.
// Cette différence est linéaire en p. (err_moy = p*1.639e-6)
// Elle ne varie pas en fonction de n et q.
// Cette erreur est sûrement dûe à différences mineurs dans la précision du stockage des flottants
// dans la mémoire RAM et VRAM (du GPU)