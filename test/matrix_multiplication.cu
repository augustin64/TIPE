#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>

#include "../src/cnn/matrix_multiplication.cu"


float random_float(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float **matrix, int n, int p) {
  for (int i=0; i < n; i++) {
    for (int j=0; j < p; j++) {
        matrix[i][j] = random_float(0.0f, 15.0f);
    }
  }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float** create_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(sizeof(float)*p);
    }

    fill_matrix_random(matrix, n, p);
    return matrix;
}


float** create_empty_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(p*sizeof(float));
        for (int j=0; j < p; j++) {
            matrix[i][j] = 0.;
        }
    }
    return matrix;
}


bool check_matrices_equality(float** m1, float** m2, int n, int p) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            if (fabs(m1[i][j] - m2[i][j]) > 0.001) {
                return false;
            }
        }
    }
    return true;
}


int main() {
    clock_t start, end;
    double cpu_time_used;

    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = check_cuda_compatibility();
    if (!cuda_compatible) {
        printf("CUDA not compatible, skipping tests.\n");
        return 0;
    }
    printf("OK\n");


    printf("Generating matrices.\n");
    srand(time(NULL));
    int n = 3;
    int p = 3;
    int q = 3;
    float** matrix1 = create_matrix(n, p);
    float** matrix2 = create_matrix(p, q);
    float** result_gpu = create_empty_matrix(n, q);
    float** result_cpu = create_empty_matrix(n, q);
    printf("OK\n");


    printf("Computing on GPU.\n");
    start = clock();
    matrix_multiplication_device(matrix1, matrix2, result_gpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time used for GPU: %lf seconds\n", cpu_time_used);
    printf("OK\n");


    printf("Computing on CPU.\n");
    start = clock();
    matrix_multiplication_host(matrix1, matrix2, result_gpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time used for CPU: %lf seconds\n", cpu_time_used);
    printf("OK\n");


    printf("Checking equality.\n");
    if (!check_matrices_equality(result_gpu, result_cpu, n, q)) {
        return 1;
    }
    printf("OK\n");
    
    return 0;
}