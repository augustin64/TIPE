#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>

#include "../cnn/include/matrix_multiplication.h"


float random_float(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float **matrix, int n, int p) {
  for (int i=0; i < n; i++) {
    for (int j=0; j < p; j++) {
        matrix[i][j] = random_float(0.0f, 15.0f);
    }
  }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float** create_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(sizeof(float)*p);
    }

    fill_matrix_random(matrix, n, p);
    return matrix;
}


float** create_empty_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(p*sizeof(float));
        for (int j=0; j < p; j++) {
            matrix[i][j] = 0.;
        }
    }
    return matrix;
}

float max_float(float a, float b) {
    return a > b ? a : b;
}


bool check_matrices_equality(float** m1, float** m2, int n, int p) {
    float err_max = 0.;
    float err_moy = 0.;
    float err_percent = 0.;
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            if (fabs(m1[i][j] - m2[i][j]) > 0.8) {
                //printf("%d %d\n", i, j);
                //return false;
            }
            err_percent = 2*fabs(m1[i][j] - m2[i][j])/fabs(m1[i][j] + m2[i][j]);
            err_max = max_float(err_max, err_percent);
            err_moy += err_percent;
        }
    }
    printf("err_max:%lf\n", err_max);
    printf("err_moy:%lf\n", err_moy/(n*p));
    return true;
}


int main(int argc, char* argv[]) {
    if (argc < 4) {
        return 1;
    }
    int n = strtol(argv[1], NULL, 10);
    int p = strtol(argv[2], NULL, 10);
    int q = strtol(argv[3], NULL, 10);

    clock_t start, end;
    double cpu_time_used;


    srand(time(NULL));
    float** matrix1 = create_matrix(n, p);
    float** matrix2 = create_matrix(p, q);
    float** result_gpu = create_empty_matrix(n, q);
    float** result_cpu = create_empty_matrix(n, q);

    start = clock();
    matrix_multiplication_device(matrix1, matrix2, result_gpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("GPU:%lf\n", cpu_time_used);
    
    start = clock();
    //matrix_multiplication_host(matrix1, matrix2, result_cpu, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("CPU:%lf\n", cpu_time_used);

    //check_matrices_equality(result_gpu, result_cpu, n, q);
    
    return 0;
}

// On obtient une différence entre le calcul fait par le GPU et par le CPU.
// Cette différence est linéaire en p. (err_moy = p*1.639e-6)
// Elle ne varie pas en fonction de n et q.
// Cette erreur est sûrement dûe à différences mineurs dans la précision du stockage des flottants
// Dans la mémoire RAM et VRAM (du GPU)