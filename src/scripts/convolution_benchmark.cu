#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <assert.h>
#include <math.h>
#include <time.h>

#include "../cnn/include/convolution.h"
#include "../cnn/include/struct.h"
#include "../include/colors.h"
#include "../include/utils.h"


float random_float(float low, float high) {
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float ***matrix, int n, int p, int q, float max_val) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            for (int k=0; k < q; k++) {
                matrix[i][j][k] = random_float(0.0f, max_val);
            }
        }
    }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float*** create_matrix(int n, int p, int q, float max_val) {
    float*** matrix = (float***)malloc(n*sizeof(float**));
    for (int i=0; i < n; i++) {
        matrix[i] = (float**)malloc(sizeof(float*)*p);
        for (int j=0; j < p; j++) {
            matrix[i][j] = (float*)malloc(sizeof(float)*q);
        }
    }

    fill_matrix_random(matrix, n, p, q, max_val);
    return matrix;
}


float*** create_empty_matrix(int n, int p, int q) {
    float*** matrix = (float***)malloc(n*sizeof(float**));
    for (int i=0; i < n; i++) {
        matrix[i] = (float**)malloc(sizeof(float*)*p);
        for (int j=0; j < p; j++) {
            matrix[i][j] = (float*)malloc(sizeof(float)*q);
            for (int k=0; k < q; k++) {
                matrix[i][j][k] = 0.;
            }
        }
    }
    return matrix;
}

void free_matrix(float*** matrix, int n, int p) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            free(matrix[i][j]);
        }
        free(matrix[i]);
    }
    free(matrix);
}

float max_float(float a, float b) {
    return a > b ? a : b;
}

bool check_matrices_equality(float*** m1, float*** m2, int n, int p, int q, int acceptation) {
    float err_max = 0.;
    float err_moy = 0.;
    float err_percent = 0.;
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            for (int k=0; k < q; k++) {
                if (fabs(m1[i][j][k] - m2[i][j][k]) > 0.01*acceptation) {
                    //printf(RED "diff %d %d %d: %f val: %f et %f\n" RESET, i, j, k, fabs(m1[i][j][k] - m2[i][j][k]), m1[i][j][k], m2[i][j][k]);
                    //return false;
                }
                err_percent = 2*fabs(m1[i][j][k] - m2[i][j][k])/fabs(m1[i][j][k] + m2[i][j][k]);
                err_max = max_float(err_max, err_percent);
                err_moy += err_percent;
            }
        }
    }
    printf("err_max:%lf\n", err_max);
    printf("err_moy:%lf\n", err_moy/(n*p*q));
    return true;
}

void run_convolution_test(int input_dim, int output_dim, int rows, int columns) {
    assert(input_dim >= output_dim);
    int k_size = input_dim - output_dim +1;

    // Génération des données aléatoires
    Kernel_cnn* kernel = (Kernel_cnn*)malloc(sizeof(Kernel_cnn));
    
    kernel->k_size = k_size;
    kernel->rows = rows;
    kernel->columns = columns;

    // bias[kernel->columns][dim_output][dim_output]
    kernel->bias = create_matrix(kernel->columns, output_dim, output_dim, 15.0f);
    kernel->d_bias = create_matrix(kernel->columns, output_dim, output_dim, 1.5f);

    // w[rows][columns][k_size][k_size]
    kernel->w = (float****)malloc(sizeof(float***)*kernel->rows);
    kernel->d_w = (float****)malloc(sizeof(float***)*kernel->rows);
    for (int i=0; i < kernel->rows; i++) {
        kernel->w[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 15.0f);
        kernel->d_w[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 1.5f);
    }

    float*** input = create_matrix(kernel->rows, input_dim, input_dim, 5.0f);
    float*** output_cpu = create_empty_matrix(kernel->columns, output_dim, output_dim);
    float*** output_gpu = create_empty_matrix(kernel->columns, output_dim, output_dim);

    //printf("(%d, %d, %d, %d) Data generation complete\n", rows, columns, input_dim, output_dim);


    // Lancement des calculs
    clock_t start, end;
    double cpu_time_used, gpu_time_used;

    start = clock();
    make_convolution_device(kernel, input, output_gpu, output_dim);
    end = clock();

    gpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("GPU: %lf\n", gpu_time_used);


    start = clock();
    make_convolution_cpu(kernel, input, output_cpu, output_dim);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("CPU: %lf\n", rows, columns, input_dim, output_dim, cpu_time_used);    

    // Vérification de l'égalité des matrices
    //printf("(%d, %d, %d, %d) Checking equality.\n", rows, columns, input_dim, output_dim);
    if (!check_matrices_equality(output_gpu, output_cpu, kernel->columns, output_dim, output_dim, kernel->k_size)) {// TODO: change acceptation
        //exit(1);
    }
    //printf(GREEN "OK\n" RESET);

    free_matrix(kernel->bias, kernel->columns, output_dim);
    free_matrix(kernel->d_bias, kernel->columns, output_dim);

    for (int i=0; i < kernel->rows; i++) {
        free_matrix(kernel->w[i], kernel->columns, kernel->k_size);
        free_matrix(kernel->d_w[i], kernel->columns, kernel->k_size);
    }
    free(kernel->w);
    free(kernel->d_w);

    free_matrix(input, kernel->rows, input_dim);
    free_matrix(output_cpu, kernel->columns, output_dim);
    free_matrix(output_gpu, kernel->columns, output_dim);
}


int main(int argc, char* argv[]) {
    if (argc < 5) {
        return 1;
    }
    int n = strtol(argv[1], NULL, 10);
    int p = strtol(argv[2], NULL, 10);
    int q = strtol(argv[3], NULL, 10);
    int r = strtol(argv[4], NULL, 10);

    /*
    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = check_cuda_compatibility();
    if (!cuda_compatible) {
        printf(RED "CUDA not compatible, skipping tests.\n" RESET);
        return 0;
    }
    */
    
    srand(time(NULL));

    run_convolution_test(n, p, q, r);
    
    return 0;
}