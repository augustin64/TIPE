#include "hip/hip_runtime.h"
//! This file uses an old implementation of convolution which uses linearised matrices
//! It is therefore not compatible nor compilable now.
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <assert.h>
#include <math.h>
#include <time.h>

#include "../cnn/include/convolution.h"
#include "../cnn/include/struct.h"
#include "../include/colors.h"
#include "../include/utils.h"


float random_float(float low, float high) {
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float ***matrix, int n, int p, int q, float max_val) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            for (int k=0; k < q; k++) {
                matrix[i][j][k] = random_float(0.0f, max_val);
            }
        }
    }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float*** create_matrix(int n, int p, int q, float max_val) {
    float*** matrix = (float***)malloc(n*sizeof(float**));
    for (int i=0; i < n; i++) {
        matrix[i] = (float**)malloc(sizeof(float*)*p);
        for (int j=0; j < p; j++) {
            matrix[i][j] = (float*)malloc(sizeof(float)*q);
        }
    }

    fill_matrix_random(matrix, n, p, q, max_val);
    return matrix;
}


float*** create_empty_matrix(int n, int p, int q) {
    float*** matrix = (float***)malloc(n*sizeof(float**));
    for (int i=0; i < n; i++) {
        matrix[i] = (float**)malloc(sizeof(float*)*p);
        for (int j=0; j < p; j++) {
            matrix[i][j] = (float*)malloc(sizeof(float)*q);
            for (int k=0; k < q; k++) {
                matrix[i][j][k] = 0.;
            }
        }
    }
    return matrix;
}

void free_matrix(float*** matrix, int n, int p) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            free(matrix[i][j]);
        }
        free(matrix[i]);
    }
    free(matrix);
}


bool check_matrices_equality(float*** m1, float*** m2, int n, int p, int q, int acceptation) {
    float err_max = 0.;
    float err_moy = 0.;
    float err_percent = 0.;
    for (int i=0; i < n; i++) {
        for (int j=0; j < p; j++) {
            for (int k=0; k < q; k++) {
                if (fabs(m1[i][j][k] - m2[i][j][k]) > 0.01*acceptation) {
                    //printf(RED "diff %d %d %d: %f val: %f et %f\n" RESET, i, j, k, fabs(m1[i][j][k] - m2[i][j][k]), m1[i][j][k], m2[i][j][k]);
                    //return false;
                }
                err_percent = 2*fabs(m1[i][j][k] - m2[i][j][k])/fabs(m1[i][j][k] + m2[i][j][k]);
                err_max = fmaxf(err_max, err_percent);
                err_moy += err_percent;
            }
        }
    }
    printf("err_max:%lf\n", err_max);
    printf("err_moy:%lf\n", err_moy/(n*p*q));
    return true;
}

void run_convolution_test(int input_dim, int output_dim, int rows, int columns) {
    assert(input_dim >= output_dim);
    int k_size = input_dim - output_dim +1;

    // Génération des données aléatoires
    Kernel_cnn* kernel = (Kernel_cnn*)malloc(sizeof(Kernel_cnn));
    
    kernel->k_size = k_size;
    kernel->rows = rows;
    kernel->columns = columns;

    // bias[kernel->columns]
    kernel->bias = (float*)malloc(kernel->columns, sizeof(float));
    kernel->d_bias = (float*)malloc(kernel->columns, sizeof(float));
    #ifdef ADAM_CNN_BIAS
    kernel->s_d_bias = (float*)malloc(kernel->columns, sizeof(float));
    kernel->v_d_bias = (float*)malloc(kernel->columns, sizeof(float));
    #endif
    for (int i=0; i<kernel->columns; i++) {
        kernel->bias[i] = random_float(0.0f, 15.0f);
        kernel->d_bias[i] = random_float(0.0f, 1.5f);
        #ifdef ADAM_CNN_BIAS
        kernel->s_d_bias[i] = random_float(0.0f, 1.5f);
        kernel->v_d_bias[i] = random_float(0.0f, 1.5f);
        #endif
    }

    // weights[rows][columns][k_size][k_size]
    kernel->weights = (float****)malloc(sizeof(float***)*kernel->rows);
    kernel->d_weights = (float****)malloc(sizeof(float***)*kernel->rows);
    #ifdef ADAM_CNN_WEIGHTS
    kernel->s_d_weights = (float****)malloc(sizeof(float***)*kernel->rows);
    kernel->v_d_weights = (float****)malloc(sizeof(float***)*kernel->rows);
    #endif
    for (int i=0; i < kernel->rows; i++) {
        kernel->weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 15.0f);
        kernel->d_weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 1.5f);
        #ifdef ADAM_CNN_WEIGHTS
        kernel->s_d_weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 1.5f);
        kernel->v_d_weights[i] = create_matrix(kernel->columns, kernel->k_size, kernel->k_size, 1.5f);
        #endif
    }

    float*** input = create_matrix(kernel->rows, input_dim, input_dim, 5.0f);
    float*** output_cpu = create_empty_matrix(kernel->columns, output_dim, output_dim);
    float*** output_gpu = create_empty_matrix(kernel->columns, output_dim, output_dim);

    //printf("(%d, %d, %d, %d) Data generation complete\n", rows, columns, input_dim, output_dim);


    // Lancement des calculs
    clock_t start, end;
    double cpu_time_used, gpu_time_used;

    start = clock();
    make_convolution_device(kernel, input, output_gpu, output_dim, 1);
    end = clock();

    gpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("GPU: %lf\n", gpu_time_used);


    start = clock();
    make_convolution_cpu(kernel, input, output_cpu, output_dim, 1);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("CPU: %lf\n", cpu_time_used);    

    // Vérification de l'égalité des matrices
    //printf("(%d, %d, %d, %d) Checking equality.\n", rows, columns, input_dim, output_dim);
    if (!check_matrices_equality(output_gpu, output_cpu, kernel->columns, output_dim, output_dim, kernel->k_size)) {// TODO: change acceptation
        //exit(1);
    }
    //printf(GREEN "OK\n" RESET);

    free(kernel->bias);
    free(kernel->d_bias);
    #ifdef ADAM_CNN_BIAS
    free(kernel->s_d_bias);
    free(kernel->v_d_bias);
    #endif

    for (int i=0; i < kernel->rows; i++) {
        free_matrix(kernel->weights[i], kernel->columns, kernel->k_size);
        free_matrix(kernel->d_weights[i], kernel->columns, kernel->k_size);
        #ifdef ADAM_CNN_WEIGHTS
        free_matrix(kernel->s_d_weights[i], kernel->columns, kernel->k_size);
        free_matrix(kernel->v_d_weights[i], kernel->columns, kernel->k_size);
        #endif
    }
    free(kernel->weights);
    free(kernel->d_weights);
    #ifdef ADAM_CNN_WEIGHTS
    free(kernel->s_d_weights);
    free(kernel->v_d_weights);
    #endif

    free_matrix(input, kernel->rows, input_dim);
    free_matrix(output_cpu, kernel->columns, output_dim);
    free_matrix(output_gpu, kernel->columns, output_dim);
}


int main(int argc, char* argv[]) {
    if (argc < 5) {
        return 1;
    }
    int n = strtol(argv[1], NULL, 10);
    int p = strtol(argv[2], NULL, 10);
    int q = strtol(argv[3], NULL, 10);
    int r = strtol(argv[4], NULL, 10);

    /*
    printf("Checking CUDA compatibility.\n");
    bool cuda_compatible = check_cuda_compatibility();
    if (!cuda_compatible) {
        printf(RED "CUDA not compatible, skipping tests.\n" RESET);
        return 0;
    }
    */
    
    srand(time(NULL));

    run_convolution_test(n, p, q, r);
    
    return 0;
}