#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>

#include "include/struct.h"
#include "../common/include/utils.h"

#include "include/config.h"


void make_convolution_cpu(Kernel_cnn* kernel, float*** input, float*** output, int output_width, int stride, int padding) {
    // c'est le kernel de input
    // input[kernel->rows][kernel_k_size + output_width-1][kernel_k_size + output_width-1]
    // output[kernel->columns][output_width][output_width]
    
    int k_columns = kernel->columns;
    int k_rows = kernel->rows;
    int max_move = kernel->k_size - padding;
    int input_width = output_width*stride - 2*padding + kernel->k_size - stride;
    float f;

    for (int i=0; i < k_columns; i++) { // filtre
        for (int j=0; j < output_width; j++) { // ligne de sortie
            for (int k=0; k < output_width; k++) { // colonne de sortie
                f = kernel->bias[i][j][k];
                for (int a=0; a < k_rows; a++) { // Canal de couleur
                    for (int b=-padding; b < max_move; b++) { // ligne du filtre
                        for (int c=-padding; c < max_move; c++) { // colonne du filtre
                            int x = (stride*j+b);
                            int y = (stride*k+c);
                            if (not_outside(x, y, 0, input_width)) {
                                f += kernel->weights[a][i][b+padding][c+padding]*input[a][x][y];
                            }
                        }
                    }
                }
                output[i][j][k] = f;
            }
        }
    }
}

#ifdef __HIPCC__

__global__ void make_convolution_kernel(float**** weights, float*** bias, int k_size, int rows, int columns, float*** input, float*** output, int output_width, int stride, int padding) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < kernel->columns
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < min(output_width, k_size)
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < min(output_width, k_size)
    int max_move = k_size - padding;
    int input_width = output_width*stride - 2*padding + k_size - stride;

    if (idx >= columns || idy >= output_width || idz >= output_width) {
        return;
    }

    float f = bias[idx][idy][idz];

    for (int a=0; a < rows; a++) {
        for (int b=-padding; b < max_move; b++) {
            for (int c=-padding; c < max_move; c++) {
                int idy_2 = idy*stride+b;
                int idz_2 = idz*stride+c;
                if (not_outside(idy_2, idz_2, 0, input_width)) {
                    f += weights[a][idx][b+padding][c+padding]*input[a][idy_2][idz_2];
                }
            }
        }
    }

    output[idx][idy][idz] = f;
}

void make_convolution_device(Kernel_cnn* kernel, float*** input, float*** output, int output_width, int stride, int padding) {
    // Make computation
    dim3 gridSize(i_div_up(kernel->columns, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    // We can't pass `kernel` directly to the CUDA kernel function
    // as it will create a 'misaligned adress' error
    make_convolution_kernel<<<gridSize, blockSize>>>(kernel->weights, kernel->bias, kernel->k_size, kernel->rows, kernel->columns, input, output, output_width, stride, padding);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

#ifdef __HIPCC__
extern "C"
#endif
void make_convolution(Kernel_cnn* kernel, float*** input, float*** output, int output_width, int stride, int padding) {
    #ifndef __HIPCC__
    make_convolution_cpu(kernel, input, output, output_width, stride, padding);
    #else
    make_convolution_device(kernel, input, output, output_width, stride, padding);
    #endif
}