#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <math.h>

#include "../common/include/colors.h"
#include "../common/include/utils.h"
#include "include/convolution.h"

#include "include/make.h"

#include "include/config.h"

#ifdef __HIPCC__
__host__ __device__
#endif

/* 
* Average Pooling
*/
#ifdef __HIPCC__
__global__ void make_average_pooling_kernel(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < output_depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < output_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < output_width
    int max_move = size - padding;
    int input_width = output_width*stride - 2*padding + size - stride;

    if (idx >= output_depth || idy >= output_width || idz >= output_width) {
        return;
    }

    int nb_elements = 0;
    float sum = 0;

    for (int a=-padding; a < max_move; a++) {
        for (int b=-padding; b < max_move; b++) {
            int idy_2 = stride*idy +a;
            int idz_2 = stride*idz +b;
            if (not_outside(idy_2, idz_2, 0, input_width)) {
                sum += input[idx][idy_2][idz_2];
                nb_elements++;
            }
        }
    }
    output[idx][idy][idz] = sum/(float)nb_elements;
}

void make_average_pooling_device(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    // Make computation
    dim3 gridSize(i_div_up(output_depth, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    make_average_pooling_kernel<<<gridSize, blockSize>>>(input, output, size, output_depth, output_width, stride, padding);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void make_average_pooling_cpu(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    // input[output_depth][output_width+size-1][output_width+size-1]
    // output[output_depth][output_width][output_width]
    int max_move = size - padding;
    int input_width = output_width*stride - 2*padding + size - stride;

    for (int i=0; i < output_depth; i++) {
        for (int j=0; j < output_width; j++) {
            for (int k=0; k < output_width; k++) {
                float sum = 0.;
                int nb_elements = 0;
                for (int a=-padding; a < max_move; a++) {
                    for (int b=-padding; b < max_move; b++) {
                        int j_2 = stride*j +a;
                        int k_2 = stride*k +b;
                        if (not_outside(j_2, k_2, 0, input_width)) {
                            sum += input[i][j_2][k_2];
                            nb_elements++;
                        }
                    }
                }
                output[i][j][k] = sum/(float)nb_elements;
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_average_pooling(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    #ifndef __HIPCC__
    make_average_pooling_cpu(input, output, size, output_depth, output_width, stride, padding);
    #else
    make_average_pooling_device(input, output, size, output_depth, output_width, stride, padding);
    #endif
}





/* 
* Max Pooling
*/
#ifdef __HIPCC__
__global__ void make_max_pooling_kernel(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < output_depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < output_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < output_width
    int input_width = output_width*stride - 2*padding + size - stride;

    if (idx >= output_depth || idy >= output_width || idz >= output_width) {
        return;
    }

    int max_move = size - padding;
    float m = -FLT_MAX;
    float temp;

    for (int a=-padding; a < max_move; a++) {
        for (int b=-padding; b < max_move; b++) {
            int idy_2 = stride*idy +a;
            int idz_2 = stride*idz +b;
            if (not_outside(idy_2, idz_2, 0, input_width)) {
                temp = input[idx][idy_2][idz_2];
                m = m > temp ? m : temp; // max(m, temp)
            }
        }
    }
    output[idx][idy][idz] = m;
}

void make_max_pooling_device(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    // Make computation
    dim3 gridSize(i_div_up(output_depth, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    make_max_pooling_kernel<<<gridSize, blockSize>>>(input, output, size, output_depth, output_width, stride, padding);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void make_max_pooling_cpu(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    // input[output_depth][output_width+size-1][output_width+size-1]
    // output[output_depth][output_width][output_width]
    int max_move = size - padding;
    int input_width = output_width*stride - 2*padding + size - stride;
    float m;
    for (int i=0; i < output_depth; i++) {
        for (int j=0; j < output_width; j++) {
            for (int k=0; k < output_width; k++) {
                m = -FLT_MAX;
                for (int a=-padding; a < max_move; a++) {
                    for (int b=-padding; b < max_move; b++) {
                        int j_2 = stride*j +a;
                        int k_2 = stride*k +b;
                        if (not_outside(j_2, k_2, 0, input_width)) {
                            m = fmaxf(m, input[i][j_2][k_2]);
                        }
                    }
                }
                output[i][j][k] = m;
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_max_pooling(float*** input, float*** output, int size, int output_depth, int output_width, int stride, int padding) {
    #ifndef __HIPCC__
    make_max_pooling_cpu(input, output, size, output_depth, output_width, stride, padding);
    #else
    make_max_pooling_device(input, output, size, output_depth, output_width, stride, padding);
    #endif
}





/*
* Dense
*/
#ifdef __HIPCC__
__global__ void make_dense_kernel(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < size_output

    if (idx >= size_output) {
        return;
    }
    float f = kernel->bias[idx];

    for (int j=0; j < size_input; j++) {
        f += kernel->weights[j][idx]*input[j];
    }
    output[idx] = f;
}

void make_dense_device(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    // Make computation
    dim3 gridSize(i_div_up(size_output, BLOCKSIZE_x*BLOCKSIZE_y), 1, 1);
    dim3 blockSize(BLOCKSIZE_x*BLOCKSIZE_y, 1, BLOCKSIZE_z);

    make_dense_kernel<<<gridSize, blockSize>>>(kernel, input, output, size_input, size_output);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

#ifdef __HIPCC__
extern "C"
#endif
void make_dense_cpu(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    // input[size_input]
    // output[size_output]
    float f;

    for (int i=0; i < size_output; i++) {
        f = kernel->bias[i];
        for (int j=0; j < size_input; j++) {
            f += kernel->weights[j][i]*input[j];
        }
        output[i] = f;
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_dense(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    #ifndef __HIPCC__
    make_dense_cpu(kernel, input, output, size_input, size_output);
    #else
    make_dense_device(kernel, input, output, size_input, size_output);
    #endif
}





/*
* Dense linearized
*/
#ifdef __HIPCC__
__global__ void make_dense_linearized_kernel(float** weights, float* bias, float*** input, float* output, int input_depth, int input_width, int size_output) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < size_output

    if (idx >= size_output) {
        return;
    }
    float f = bias[idx];

    for (int i=0; i < input_depth; i++) {
        for (int j=0; j < input_width; j++) {
            for (int k=0; k < input_width; k++) {
                f += input[i][j][k]*weights[k + j*input_width + i*input_depth][idx];
            }
        }
    }
    output[idx] = f;
}

void make_dense_linearized_device(Kernel_nn* kernel, float*** input, float* output, int input_depth, int input_width, int size_output) {
    // Make computation
    dim3 gridSize(i_div_up(size_output, BLOCKSIZE_x*BLOCKSIZE_y), 1, 1);
    dim3 blockSize(BLOCKSIZE_x*BLOCKSIZE_y, 1, BLOCKSIZE_z);

    make_dense_linearized_kernel<<<gridSize, blockSize>>>(kernel->weights, kernel->bias, input, output, input_depth, input_width, size_output);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void make_dense_linearized_cpu(Kernel_nn* kernel, float*** input, float* output, int input_depth, int input_width, int size_output) {
    // input[input_depth][input_width][input_width]
    // output[size_output]
    float f;

    for (int l=0; l < size_output; l++) {
        f = kernel->bias[l];
        for (int i=0; i < input_depth; i++) {
            for (int j=0; j < input_width; j++) {
                for (int k=0; k < input_width; k++) {
                    f += input[i][j][k]*kernel->weights[k + j*input_width + i*input_depth][l];
                }
            }
        }
        output[l] = f;
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_dense_linearized(Kernel_nn* kernel, float*** input, float* output, int input_depth, int input_width, int size_output) {
    #ifndef __HIPCC__
    make_dense_linearized_cpu(kernel, input, output, input_depth, input_width, size_output);
    #else
    make_dense_linearized_device(kernel, input, output, input_depth, input_width, size_output);
    #endif
}
