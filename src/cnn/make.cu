#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "include/convolution.h"
#include "../include/colors.h"
#include "../include/utils.h"

#include "include/make.h"

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 8
#define BLOCKSIZE_z 8

float max_flt(float a, float b) {
    // Return the max between the two floats
    if (a > b) {
        return a;
    }
    return b;
}





/* 
* Average Pooling
*/
#ifdef __HIPCC__
__global__ void make_average_pooling_kernel(float*** input, float*** output, int size, int output_depth, int output_width) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < output_depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < output_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < output_width
    int n = size*size;

    if (idx >= output_depth || idy >= output_width || idz >= output_width) {
        return;
    }

    float sum = 0;

    for (int a=0; a < size; a++) {
        for (int b=0; b < size; b++) {
            sum += input[idx][size*idy +a][size*idz +b];
        }
    }
    output[idx][idy][idz] = sum/(float)n;
}

void make_average_pooling_device(float*** input, float*** output, int size, int output_depth, int output_width) {
    // Make computation
    dim3 gridSize(i_div_up(output_depth, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    make_average_pooling_kernel<<<gridSize, blockSize>>>(input, output, size, output_depth, output_width);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void make_average_pooling_cpu(float*** input, float*** output, int size, int output_depth, int output_width) {
    // input[output_depth][output_width+size-1][output_width+size-1]
    // output[output_depth][output_width][output_width]
    float sum;
    int n = size*size;

    for (int i=0; i < output_depth; i++) {
        for (int j=0; j < output_width; j++) {
            for (int k=0; k < output_width; k++) {
                sum = 0;
                for (int a=0; a < size; a++) {
                    for (int b=0; b < size; b++) {
                        sum += input[i][size*j +a][size*k +b];
                    }
                }
                output[i][j][k] = sum/(float)n;
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_average_pooling(float*** input, float*** output, int size, int output_depth, int output_width) {
    #ifndef __HIPCC__
    make_average_pooling_cpu(input, output, size, output_depth, output_width);
    #else
    make_average_pooling_device(input, output, size, output_depth, output_width);
    #endif
}





/* 
* Max Pooling
*/
#ifdef __HIPCC__
__global__ void make_max_pooling_kernel(float*** input, float*** output, int size, int output_depth, int output_width) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < output_depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < output_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < output_width

    if (idx >= output_depth || idy >= output_width || idz >= output_width) {
        return;
    }

    float m = FLT_MIN;
    float temp;

    for (int a=0; a < size; a++) {
        for (int b=0; b < size; b++) {
            temp = input[idx][size*idy +a][size*idz +b];
            m = m > temp ? m : temp; // max(m, temp)
        }
    }
    output[idx][idy][idz] = m;
}

void make_max_pooling_device(float*** input, float*** output, int size, int output_depth, int output_width) {
    // Make computation
    dim3 gridSize(i_div_up(output_depth, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    make_max_pooling_kernel<<<gridSize, blockSize>>>(input, output, size, output_depth, output_width);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void make_max_pooling_cpu(float*** input, float*** output, int size, int output_depth, int output_width) {
    // input[output_depth][output_width+size-1][output_width+size-1]
    // output[output_depth][output_width][output_width]
    float m;
    for (int i=0; i < output_depth; i++) {
        for (int j=0; j < output_width; j++) {
            for (int k=0; k < output_width; k++) {
                m = FLT_MIN;
                for (int a=0; a < size; a++) {
                    for (int b=0; b < size; b++) {
                        m = max_flt(m, input[i][size*j +a][size*k +b]);
                    }
                }
                output[i][j][k] = m;
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_max_pooling(float*** input, float*** output, int size, int output_depth, int output_width) {
    #ifndef __HIPCC__
    make_max_pooling_cpu(input, output, size, output_depth, output_width);
    #else
    make_max_pooling_device(input, output, size, output_depth, output_width);
    #endif
}





/*
* Dense
*/
#ifdef __HIPCC__
__global__ void make_dense_kernel(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < size_output

    if (idx >= size_output) {
        return;
    }
    float f = kernel->bias[idx];

    for (int j=0; j < size_input; j++) {
        f += kernel->weights[j][idx]*input[j];
    }
    output[idx] = f;
}

void make_dense_device(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    // Make computation
    dim3 gridSize(i_div_up(size_output, BLOCKSIZE_x*BLOCKSIZE_y), 1, 1);
    dim3 blockSize(BLOCKSIZE_x*BLOCKSIZE_y, 1, BLOCKSIZE_z);

    make_dense_kernel<<<gridSize, blockSize>>>(kernel, input, output, size_input, size_output);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

#ifdef __HIPCC__
extern "C"
#endif
void make_dense_cpu(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    // input[size_input]
    // output[size_output]
    float f;

    for (int i=0; i < size_output; i++) {
        f = kernel->bias[i];
        for (int j=0; j < size_input; j++) {
            f += kernel->weights[j][i]*input[j];
        }
        output[i] = f;
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_dense(Kernel_nn* kernel, float* input, float* output, int size_input, int size_output) {
    #ifndef __HIPCC__
    make_dense_cpu(kernel, input, output, size_input, size_output);
    #else
    make_dense_device(kernel, input, output, size_input, size_output);
    #endif
}





/*
* Dense linearized
*/
#ifdef __HIPCC__
__global__ void make_dense_linearized_kernel(float** weights, float* bias, float*** input, float* output, int depth_input, int dim_input, int size_output) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < size_output

    if (idx >= size_output) {
        return;
    }
    float f = bias[idx];

    for (int i=0; i < depth_input; i++) {
        for (int j=0; j < dim_input; j++) {
            for (int k=0; k < dim_input; k++) {
                f += input[i][j][k]*weights[k + j*dim_input + i*depth_input][idx];
            }
        }
    }
    output[idx] = f;
}

void make_dense_linearized_device(Kernel_nn* kernel, float*** input, float* output, int depth_input, int dim_input, int size_output) {
    // Make computation
    dim3 gridSize(i_div_up(size_output, BLOCKSIZE_x*BLOCKSIZE_y), 1, 1);
    dim3 blockSize(BLOCKSIZE_x*BLOCKSIZE_y, 1, BLOCKSIZE_z);

    make_dense_linearized_kernel<<<gridSize, blockSize>>>(kernel->weights, kernel->bias, input, output, depth_input, dim_input, size_output);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void make_dense_linearized_cpu(Kernel_nn* kernel, float*** input, float* output, int depth_input, int dim_input, int size_output) {
    // input[depth_input][dim_input][dim_input]
    // output[size_output]
    float f;

    for (int l=0; l < size_output; l++) {
        f = kernel->bias[l];
        for (int i=0; i < depth_input; i++) {
            for (int j=0; j < dim_input; j++) {
                for (int k=0; k < dim_input; k++) {
                    f += input[i][j][k]*kernel->weights[k + j*dim_input + i*depth_input][l];
                }
            }
        }
        output[l] = f;
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void make_dense_linearized(Kernel_nn* kernel, float*** input, float* output, int depth_input, int dim_input, int size_output) {
    #ifndef __HIPCC__
    make_dense_linearized_cpu(kernel, input, output, depth_input, dim_input, size_output);
    #else
    make_dense_linearized_device(kernel, input, output, depth_input, dim_input, size_output);
    #endif
}
