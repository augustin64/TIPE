#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>

#include "../include/colors.h"
#include "../include/utils.h"

#include "include/function.h"

#include "include/config.h"

//* Identity
#ifdef __HIPCC__
__device__ float device_identity(float x) {
    return x;
}

__device__ float device_identity_derivative(float x) {
    (void)x;
    return 1;
}
#endif

float identity(float x) {
    return x;
}

float identity_derivative(float x) {
    (void)x;
    return 1;
}


//* Sigmoid
#ifdef __HIPCC__
__device__ float device_sigmoid(float x) {
    return 1/(1 + exp(-x));
}

__device__ float device_sigmoid_derivative(float x) {
    float tmp = exp(-x);
    return tmp/((1+tmp)*(1+tmp));
}
#endif

float sigmoid(float x) {
    return 1/(1 + exp(-x));
}

float sigmoid_derivative(float x) {
    float tmp = exp(-x);
    return tmp/((1+tmp)*(1+tmp));
}


//* RELU
#ifdef __HIPCC__
__device__ float device_relu(float x) {
    return fmaxf(0, fminf(x, RELU_CLIP_VALUE));
}

__device__ float device_relu_derivative(float x) {
    if (x > 0)
        return 1;
    return 0;
}
#endif

float relu(float x) {
    return fmaxf(0, fminf(x, RELU_CLIP_VALUE));
}

float relu_derivative(float x) {
    if (x > 0)
        return 1;
    return 0;
}


//* Leaky RELU
#ifdef __HIPCC__
__device__ float device_leaky_relu(float x) {
    if (x>0)
        return fminf(x, RELU_CLIP_VALUE);
    return x*LEAKER;
}

__device__ float device_leaky_relu_derivative(float x) {
    if (x > 0)
        return 1;
    return LEAKER;
}
#endif

float leaky_relu(float x) {
    if (x>0)
        return fminf(x, RELU_CLIP_VALUE);
    return x*LEAKER;
}

float leaky_relu_derivative(float x) {
    if (x > 0)
        return 1;
    return LEAKER;
}


//* Tanh
#ifdef __HIPCC__
__device__ float device_tanh_(float x) {
    return tanh(x);
}

__device__ float device_tanh_derivative(float x) {
    float a = tanh(x);
    return 1 - a*a;
}
#endif

float tanh_(float x) {
    return tanh(x);
}

float tanh_derivative(float x) {
    float a = tanh(x);
    return 1 - a*a;
}




#ifdef __HIPCC__
/*
 * Définition des pointeurs de fonctions pour CUDA
 * voir https://stackoverflow.com/a/15646771
*/
__device__ funcPtr ptr_sigmoid = device_sigmoid;
__device__ funcPtr ptr_relu = device_relu;
__device__ funcPtr ptr_leaky_relu = device_leaky_relu;
__device__ funcPtr ptr_tanh = device_tanh_;
__device__ funcPtr ptr_identity = device_identity;

__device__ funcPtr ptr_identity_derivative = device_identity_derivative;
__device__ funcPtr ptr_sigmoid_derivative = device_sigmoid_derivative;
__device__ funcPtr ptr_relu_derivative = device_relu_derivative;
__device__ funcPtr ptr_leaky_relu_derivative = device_leaky_relu_derivative;
__device__ funcPtr ptr_tanh_derivative = device_tanh_derivative;
#endif



void apply_softmax_input(float ***input, int depth, int rows, int columns) {
    float m = -FLT_MAX;
    float sum=0;
    for (int i=0; i < depth; i++) {
        for (int j=0; j < rows; j++) {
            for (int k=0; k < columns; k++) {
                m = fmaxf(m, input[i][j][k]);
            }
        }
    }
    for (int i=0; i < depth; i++) {
        for (int j=0; j < rows; j++) {
            for (int k=0; k < columns; k++) {
                input[i][j][k] = exp(m-input[i][j][k]);
                sum += input[i][j][k];
            }
        }
    }
    for (int i=0; i < depth; i++) {
        for (int j=0; j < rows; j++) {
            for (int k=0; k < columns; k++) {
                input[i][j][k] = input[i][j][k]/sum;
            }
        }
    }
}


/* 
* Apply function on input
*/
#ifdef __HIPCC__
__global__ void apply_function_input_kernel(funcPtr f, float*** input, int depth, int rows, int columns) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < rows
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < columns

    if (idx >= depth || idy >= rows || idz >= columns) {
        return;
    }

    input[idx][idy][idz] = (*f)(input[idx][idy][idz]);
}


void apply_function_input_device(int activation, float*** input, int depth, int rows, int columns) {
    // Make computation
    dim3 gridSize(i_div_up(depth, BLOCKSIZE_x), i_div_up(rows, BLOCKSIZE_y), i_div_up(columns, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    funcPtr activation_function = get_activation_function_cuda(activation);

    apply_function_input_kernel<<<gridSize, blockSize>>>(activation_function, input, depth, rows, columns);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void apply_function_input_cpu(int activation, float*** input, int depth, int rows, int columns) {
    funcPtr f = get_activation_function(activation);

    for (int i=0; i < depth; i++) {
        for (int j=0; j < rows; j++) {
            for (int k=0; k < columns; k++) {
                input[i][j][k] = (*f)(input[i][j][k]);
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void apply_function_input(int activation, float*** input, int depth, int rows, int columns) {
    #ifndef __HIPCC__
    apply_function_input_cpu(activation, input, depth, rows, columns);
    #else
    apply_function_input_device(activation, input, depth, rows, columns);
    #endif
}

void apply_function_to_matrix(int activation, float*** input, int depth, int dim) {
    if (activation == SOFTMAX) {
        return apply_softmax_input(input, depth, dim, dim);
    }
    if (activation >= 1) { // Exclude negative values (derivative)
        return apply_function_input(activation, input, depth, dim, dim);
    }
    printf_error((char*)"fonction d'activation inconnue (apply_function_to_matrix): ");
    printf("%d\n", activation);
}


void apply_function_to_vector(int activation, float*** input, int dim) {
    if (activation == SOFTMAX) {
        return apply_softmax_input(input, 1, 1, dim);
    }
    if (activation >= 1) { // Exclude negative values (derivative)
        return apply_function_input(activation, input, 1, 1, dim);
    }
    printf_error((char*)"fonction d'activation inconnue (apply_function_to_vector): ");
    printf("%d\n", activation);
}


funcPtr get_activation_function(int activation) {
    switch (activation) {
        case RELU:
            return &relu;
        case -RELU:
            return &relu_derivative;

        case IDENTITY:
            return &identity;
        case -IDENTITY:
            return &identity_derivative;

        case SIGMOID:
            return &sigmoid;
        case -SIGMOID:
            return &sigmoid_derivative;
        
        case LEAKY_RELU:
            return &leaky_relu;
        case -LEAKY_RELU:
            return &leaky_relu_derivative;

        case TANH:
            return &tanh_;
        case -TANH:
            return &tanh_derivative;

        case SOFTMAX:
            printf_error((char*)"impossible de renvoyer la fonction softmax\n");
            return NULL;
        case -SOFTMAX:
            printf_error((char*)"impossible de renvoyer la dérivée de la fonction softmax\n");
            return NULL;

        default:
            printf_error((char*)"fonction d'activation inconnue (get_activation_function_cuda): ");
            printf("%d\n", activation);
            return NULL;
    }
}


#ifdef __HIPCC__
extern "C"
funcPtr get_activation_function_cuda(int activation) {
    funcPtr host_function;
    
    switch (activation) {
        case RELU:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_relu), sizeof(funcPtr)));
            break;
        case -RELU:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_relu_derivative), sizeof(funcPtr)));
            break;

        case IDENTITY:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_identity), sizeof(funcPtr)));
            break;
        case -IDENTITY:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_identity_derivative), sizeof(funcPtr)));
            break;

        case SIGMOID:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_sigmoid), sizeof(funcPtr)));
            break;
        case -SIGMOID:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_sigmoid_derivative), sizeof(funcPtr)));
            break;
        
        case LEAKY_RELU:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_leaky_relu), sizeof(funcPtr)));
            break;
        case -LEAKY_RELU:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_leaky_relu_derivative), sizeof(funcPtr)));
            break;

        case TANH:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_tanh), sizeof(funcPtr)));
            break;
        case -TANH:
            gpuErrchk( hipMemcpyFromSymbol(&host_function, HIP_SYMBOL(ptr_tanh_derivative), sizeof(funcPtr)));
            break;

        case SOFTMAX:
            printf_error((char*)"impossible de renvoyer la fonction softmax\n");
            return NULL;
        case -SOFTMAX:
            printf_error((char*)"impossible de renvoyer la dérivée de la fonction softmax\n");
            return NULL;

        default:
            printf_error((char*)"fonction d'activation inconnue (get_activation_function_cuda): ");
            printf("%d\n", activation);
            return NULL;
    }
    return host_function;
}
#endif