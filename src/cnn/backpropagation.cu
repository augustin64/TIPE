#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <math.h>

#include "include/backpropagation.h"
#include "../common/include/colors.h"
#include "../common/include/utils.h"
#include "include/struct.h"

#include "include/config.h"


int not_outside(int x, int y, int lower_bound, int upper_bound) {
    return !(x < lower_bound || y < lower_bound || x >= upper_bound || y>= upper_bound);
}


/*
* Softmax backward MSE
*/
#ifdef __HIPCC__
__global__ void softmax_backward_mse_kernel(float* input, float* output, int size) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;

    if (idx >= size) {
        return;
    }

    int input_val = input[idx];
    int output_val = output[idx];

    input[idx] = (output_val-input_val)*input_val*(1-input_val);
}

void softmax_backward_mse_device(float* input, float* output, int size) {
    // Make computation
    dim3 gridSize(i_div_up(size, BLOCKSIZE_x));
    dim3 blockSize(BLOCKSIZE_x);

    softmax_backward_mse_kernel<<<gridSize, blockSize>>>(input, output, size);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void softmax_backward_mse_cpu(float* input, float* output, int size) {
    /* Input et output ont la même taille */

    for (int i=0; i < size; i++){
        input[i] = (output[i]-input[i])*input[i]*(1-input[i]);
    }
}

void softmax_backward_mse(float* input, float* output, int size) {
    #ifdef __HIPCC__
    softmax_backward_mse_device(input, output, size);
    #else
    softmax_backward_mse_cpu(input, output, size);
    #endif
}


/*
* Softmax backward Cross entropy
*/
#ifdef __HIPCC__
__global__ void softmax_backward_cross_entropy_kernel(float* input, float* output, int size) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;

    if (idx >= size) {
        return;
    }

    input[idx] = output[idx] - input[idx];
}

void softmax_backward_cross_entropy_device(float* input, float* output, int size) {
    // Make computation
    dim3 gridSize(i_div_up(size, BLOCKSIZE_x));
    dim3 blockSize(BLOCKSIZE_x);

    softmax_backward_cross_entropy_kernel<<<gridSize, blockSize>>>(input, output, size);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void softmax_backward_cross_entropy_cpu(float* input, float* output, int size) {
    /* Input et output ont la même taille */

    for (int i=0; i < size; i++){
        input[i] = output[i] - input[i];
    }
}

void softmax_backward_cross_entropy(float* input, float* output, int size) {
    #ifdef __HIPCC__
    softmax_backward_cross_entropy_device(input, output, size);
    #else
    softmax_backward_cross_entropy_cpu(input, output, size);
    #endif
}


/*
* Backward average pooling
*/
#ifdef __HIPCC__
__global__ void backward_average_pooling_kernel(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < output_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < output_width

    if (idx >= depth || idy >= output_width || idz >= output_width) {
        return;
    }
    int max_move = kernel_size - padding;

    for (int a=-padding; a < max_move; a++) {
        for (int b=-paddding; b < max_move; b++) {
            int idy_2 = stride*idy +a;
            int idz_2 = stride*idz +b:
            if (not_outside(idy_2, idz_2, 0, input_width)) {
                int y = min(idy_2+1, min(kernel_size, input_width - idy_2));
                int z = min(idz_2+1, min(kernel_size, input_width - idz_2));
                input[idx][idy_2][idz_2] += output[idx][idy][idz]/(y*z);
            }
        }
    }
}


void backward_average_pooling_device(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    // Make computation
    dim3 gridSize(i_div_up(depth, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    reset_3d_array(input, depth, input_width, input_width);

    backward_average_pooling_kernel<<<gridSize, blockSize>>>(input, output, input_width, output_width, depth, kernel_size, stride, padding);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void backward_average_pooling_cpu(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    /* Input et output ont la même profondeur (depth) */

    reset_3d_array(input, depth, input_width, input_width);
    int max_move = kernel_size - padding;

    for (int i=0; i < depth; i++) {
        for (int j=0; j < output_width; j++) {
            for (int k=0; k < output_width; k++) {
                for (int a=-padding; a < max_move; a++) {
                    for (int b=-padding; b < max_move; b++) {
                        int j_2 = stride*j +a;
                        int k_2 = stride*k + b;
                        if (not_outside(j_2, k_2, 0, input_width)){
                            int j_3 = min(j_2+1, min(kernel_size, input_width - j_2));
                            int k_3 = min(k_2+1, min(kernel_size, input_width - k_2));
                            input[i][j_2][k_2] += output[i][j][k]/(j_3*k_3);
                        }
                    }
                }
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void backward_average_pooling(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    #ifndef __HIPCC__
    backward_average_pooling_cpu(input, output, input_width, output_width, depth, kernel_size, stride, padding);
    #else
    backward_average_pooling_device(input, output, input_width, output_width, depth, kernel_size, stride, padding);
    #endif
}


/*
* Backward max pooling
*/
#ifdef __HIPCC__
__global__ void backward_max_pooling_kernel(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < output_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < output_width

    if (idx >= depth || idy >= output_width || idz >= output_width) {
        return;
    }
    int max_move = kernel_size - padding;
    float m = -FLT_MAX;
    int a_max = -1;
    int b_max = -1;
    int cpt = 0;

    for (int a=-padding; a < max_move; a++) {
        for (int b=-padding; b < max_move; b++) {
            int idy_2 = stride*idy +a;
            int idz_2 = stride*idz +b;
            if (not_outside(idy_2, idz_2, 0, input_width)) {
                if (input[idx][idy_2][idz_2] > m) {
                    m = input[idx][idy_2][idz_2];
                    a_max = a;
                    b_max = b;
                }
                input[idx][idy_2][idz_2] = 0;
                cpt++;
            }
        }
    }
    if (cpt==0) {
        printf_error("Dimensions ou stride ou padding erroné dans 'backward_max_pooling_cpu'\n");
    }
    input[idx][stride*idy +a_max][stride*idz +b_max] = output[idx][idy][idz]/cpt;
}


void backward_max_pooling_device(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    // Make computation
    dim3 gridSize(i_div_up(depth, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_z));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    backward_max_pooling_kernel<<<gridSize, blockSize>>>(input, output, input_width, output_width, depth, kernel_size, stride, padding);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void backward_max_pooling_cpu(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    float m; // Maximum
    int a_max, b_max; // Indices du maximum
    int cpt;
    int max_move = kernel_size - padding;

    for (int i=0; i < depth; i++) {
        for (int j=0; j < output_width; j++) {
            for (int k=0; k < output_width; k++) {
                m = -FLT_MAX;
                a_max = -1;
                b_max = -1;
                cpt = 0;

                for (int a=-padding; a < max_move; a++) {
                    for (int b=-padding; b < max_move; b++) {
                        int j_2 = stride*j +a;
                        int k_2 = stride*k +b;
                        if (not_outside(j_2, k_2, 0, input_width)) {
                            if (input[i][j_2][k_2] > m) {
                                m = input[i][j_2][k_2];
                                a_max = a;
                                b_max = b;
                            }
                            input[i][j_2][k_2] = 0;
                            cpt++;
                        }
                    }
                }
                if (cpt==0) {
                    printf_error("Dimensions ou stride ou padding erroné dans 'backward_max_pooling_cpu'\n");
                }
                else {
                    input[i][stride*j +a_max][stride*k +b_max] = output[i][j][k]/cpt;
                }
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void backward_max_pooling(float*** input, float*** output, int input_width, int output_width, int depth, int kernel_size, int stride, int padding) {
    #ifndef __HIPCC__
    backward_max_pooling_cpu(input, output, input_width, output_width, depth, kernel_size, stride, padding);
    #else
    backward_max_pooling_device(input, output, input_width, output_width, kernel_size, depth, stride, padding);
    #endif
}

/*
* Backward Dense
*/
#ifdef __HIPCC__
__global__ void backward_dense_kernel_1(Kernel_nn* ker, float* input, float* output, int size_input, int size_output) {
    // Équivalents respectifs de i, j et k dans la boucle effectuée par le cpu
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < size_input
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < size_output

    if (idx >= size_input || idy >= size_output) {
        return;
    }

    if (idx == 0) {
        ker->d_bias[idy] += output[idy];
    }
    ker->d_weights[idx][idy] += input[idx]*output[idy];
}

__global__ void backward_dense_kernel_2(float** weights, float* input, float* input_z, float* output, int size_input, int size_output, funcPtr d_f) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < size_input

    if (idx >= size_input) {
        return;
    }

    float tmp=0;
    for (int j=0; j < size_output; j++) {
        tmp += output[j]*weights[idx][j];
    }
    input[idx] = tmp*( (*d_f)(input_z[idx]) );
}

void backward_dense_device(Kernel_nn* ker, float* input, float* input_z, float* output, int size_input, int size_output, int activation, int is_first) {
    // Make computation
    dim3 gridSize1(i_div_up(size_input, BLOCKSIZE_x), i_div_up(size_output, BLOCKSIZE_y));
    dim3 blockSize1(BLOCKSIZE_x, BLOCKSIZE_y);

    backward_dense_kernel_1<<<gridSize1, blockSize1>>>(ker, input, output, size_input, size_output);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Second kernel
    if (is_first != 1) {
        dim3 gridSize1(i_div_up(size_input, BLOCKSIZE_x));
        dim3 blockSize1(BLOCKSIZE_x);

        funcPtr d_function = get_activation_function_cuda(activation);

        backward_dense_kernel_2<<<gridSize1, blockSize1>>>(ker->weights, input, input_z, output, size_input, size_output, d_function);
        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
}
#endif

void backward_dense_cpu(Kernel_nn* ker, float* input, float* input_z, float* output, int size_input, int size_output, int activation, int is_first) {

    funcPtr d_function = get_activation_function(activation);
    // Bias
    for (int j=0; j < size_output; j++) {
        ker->d_bias[j] += output[j];
    }

    // Weights
    for (int i=0; i < size_input; i++) {
        for (int j=0; j < size_output; j++) {
            ker->d_weights[i][j] += input[i]*output[j];
        }
    }

    // Input
    if (is_first==1) {// Pas besoin de backpropager dans l'input
        return;
    }

    for (int i=0; i < size_input; i++) {
        float tmp=0;
        for (int j=0; j < size_output; j++) {
            tmp += output[j]*ker->weights[i][j];
        }
        input[i] = tmp*d_function(input_z[i]);
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void backward_dense(Kernel_nn* ker, float* input, float* input_z, float* output, int size_input, int size_output, int activation, int is_first) {
    #ifndef __HIPCC__
    backward_dense_cpu(ker, input, input_z, output, size_input, size_output, activation, is_first);
    #else
    backward_dense_device(ker, input, input_z, output, size_input, size_output, activation, is_first);
    #endif
}



/*
* Backward linearisation
*/
#ifdef __HIPCC__
__global__ void backward_linearisation_kernel_1(Kernel_nn* ker, float*** input, float* output, int input_depth, int input_width, int size_output) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < input_depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < input_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < input_width

    if (idx >= input_depth || idy >= input_width || idz >= input_width) {
        return;
    }

    int id = idx*input_width*input_width + idy*input_width + idz;
    
    for (int j=0; j < size_output; j++) {
        ker->d_weights[id][j] += input[idx][idy][idz]*output[j];
    }
    if (id == 0) {
        for (int j=0; j < size_output;  j++) {
            ker->d_bias[j] += output[j];
        }
    }
}

__global__ void backward_linearisation_kernel_2(Kernel_nn* ker, float*** input, float*** input_z, float* output, int input_depth, int input_width, int size_output, funcPtr d_f) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x; // < input_depth
    int idy = threadIdx.y + blockDim.y*blockIdx.y; // < input_width
    int idz = threadIdx.z + blockDim.z*blockIdx.z; // < input_width

    if (idx >= input_depth || idy >= input_width || idz >= input_width) {
        return;
    }
    int id = idx*input_width*input_width + idy*input_width + idz;

    float tmp=0;
    for (int j=0; j < size_output; j++) {
        tmp += output[j]*ker->weights[id][j];
    }
    input[idx][idy][idz] = tmp*( (*d_f)(input_z[idx][idy][idz]) );
}

void backward_linearisation_device(Kernel_nn* ker, float*** input, float*** input_z, float* output, int input_depth, int input_width, int size_output, int activation) {
    // Make computation
    dim3 gridSize(i_div_up(input_depth, BLOCKSIZE_x), i_div_up(input_width, BLOCKSIZE_y), i_div_up(input_width, BLOCKSIZE_y));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    backward_linearisation_kernel_1<<<gridSize, blockSize>>>(ker, input, output, input_depth, input_width, size_output);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Second kernel
    funcPtr d_function = get_activation_function_cuda(activation);

    backward_linearisation_kernel_2<<<gridSize, blockSize>>>(ker, input, input_z, output, input_depth, input_width, size_output, d_function);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif

void backward_linearisation_cpu(Kernel_nn* ker, float*** input, float*** input_z, float* output, int input_depth, int input_width, int size_output, int activation) {
   
    funcPtr d_function = get_activation_function(activation);

    // Bias
    for (int j=0; j < size_output; j++) {
        ker->d_bias[j] += output[j];
    }

    // Weights
    int cpt = 0;
    for (int i=0; i < input_depth; i++) {
        for (int k=0; k < input_width; k++) {
            for (int l=0; l < input_width; l++) {
                for (int j=0; j < size_output; j++) {
                    ker->d_weights[cpt][j] += input[i][k][l]*output[j];
                }
                cpt++;
            }
        }
    }

    // Input
    cpt = 0;
    for (int i=0; i < input_depth; i++) {
        for (int k=0; k < input_width; k++) {
            for (int l=0; l < input_width; l++) {
                float tmp=0;
                for (int j=0; j < size_output; j++) {
                    tmp += output[j]*ker->weights[cpt][j];
                }
                input[i][k][l] = tmp*d_function(input_z[i][k][l]);
                cpt++;
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void backward_linearisation(Kernel_nn* ker, float*** input, float*** input_z, float* output, int input_depth, int input_width, int size_output, int activation) {
    #ifndef __HIPCC__
    backward_linearisation_cpu(ker, input, input_z, output, input_depth, input_width, size_output, activation);
    #else
    backward_linearisation_device(ker, input, input_z, output, input_depth, input_width, size_output, activation);
    #endif
}

/*
* Backward convolution
*/
#ifdef __HIPCC__
__global__ void backward_convolution_dbias_kernel(Kernel_cnn* ker, float*** output, int output_depth, int output_width) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    int idy = threadIdx.y + blockDim.y*blockIdx.y;
    int idz = threadIdx.z + blockDim.z*blockIdx.z;
    
    if (idx >= output_depth || idy >= output_width || idz >= output_width) {
        return;
    }
    ker->d_bias[idx][idy][idz] += output[idx][idy][idz];
}

__global__ void backward_convolution_dweight_kernel(Kernel_cnn* ker, float*** input, float*** output, int input_depth, int output_depth, int output_width, int kernel_size) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    int idy = threadIdx.y + blockDim.y*blockIdx.y;
    int idz = threadIdx.z + blockDim.z*blockIdx.z;

    int idz1 = idz / kernel_size;
    int idz2 = idz % kernel_size;
    
    if (idx >= input_depth || idy >= output_depth || idz1 >= kernel_size || idz2 >= kernel_size) {
        return;
    }
    
    float tmp = 0;
    for (int l=0; l < output_width; l++) {
        for (int m=0; m < output_width; m++) {
            tmp += input[idx][l+idz1][m+idz2]*output[idy][l][m];
        }
    }
    ker->d_weights[idx][idy][idz1][idz2] += tmp;
}

__global__ void backward_convolution_propagate_kernel(Kernel_cnn* ker, float*** input, float*** input_z, float*** output, int input_depth, int input_width, int output_depth, int k_size, funcPtr d_f) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    int idy = threadIdx.y + blockDim.y*blockIdx.y;
    int idz = threadIdx.z + blockDim.z*blockIdx.z;

    if (idx >= input_depth || idy >= input_width || idz >= input_width) {
        return;
    }

    int min_m, max_m, min_n, max_n;
    float tmp = 0;
    for (int l=0; l < output_depth; l++) {
        min_m = max(0, k_size-1-idy);
        max_m = min(k_size, input_width - idy);
        min_n = max(0, k_size-1-idz);
        max_n = min(k_size, input_width-idz);
        for (int m=min_m; m < max_m; m++) {
            for (int n=min_n; n < max_n; n++) {
                tmp += output[l][idy-k_size+m+1][idz-k_size+n+1]*ker->weights[idx][l][m][n];
            }
        }
    }
    input[idx][idy][idz] = tmp*( (*d_f)(input_z[idx][idy][idz]) );
}

void backward_convolution_device(Kernel_cnn* kernel, float*** input, float*** input_z, float*** output, int input_depth, int input_width, int output_depth, int output_width, int activation, int is_first, int kernel_size, int padding, int stride) {
    // Bias Kernel
    dim3 gridSize1(i_div_up(output_depth, BLOCKSIZE_x), i_div_up(output_width, BLOCKSIZE_y), i_div_up(output_width, BLOCKSIZE_y));
    dim3 blockSize1(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    backward_convolution_dbias_kernel<<<gridSize1, blockSize1>>>(kernel, output, output_depth, output_width);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    dim3 gridSize2(i_div_up(input_depth, BLOCKSIZE_x), i_div_up(output_depth, BLOCKSIZE_y), i_div_up(kernel_size*kernel_size, BLOCKSIZE_y));
    dim3 blockSize2(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

    backward_convolution_dweight_kernel<<<gridSize2, blockSize2>>>(kernel, input, output, input_depth, output_depth, output_width, kernel_size);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // input propagation Kernel
    if (is_first != 1) {
        dim3 gridSize3(i_div_up(input_depth, BLOCKSIZE_x), i_div_up(input_width, BLOCKSIZE_y), i_div_up(input_width, BLOCKSIZE_y));
        dim3 blockSize3(BLOCKSIZE_x, BLOCKSIZE_y, BLOCKSIZE_z);

        funcPtr d_function = get_activation_function_cuda(activation);

        backward_convolution_propagate_kernel<<<gridSize3, blockSize3>>>(kernel, input, input_z, output, input_depth, input_width, output_depth, kernel_size, d_function);
    
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
}
#endif


void backward_convolution_cpu(Kernel_cnn* ker, float*** input, float*** input_z, float*** output, int input_depth, int input_width, int output_depth, int output_width, int activation, int is_first, int kernel_size, int padding, int stride) {
    
    funcPtr d_function = get_activation_function(activation);
    int max_move = kernel_size - padding;

    // Bias
    for (int i=0; i < output_depth; i++) {
        for (int j=0; j < output_width; j++) {
            for (int k=0; k < output_width; k++) {
                ker->d_bias[i][j][k] += output[i][j][k];
            }
        }
    }

    // Weights    
    for (int h=0; h < input_depth; h++) {
        for (int i=0; i < output_depth; i++) {
            for (int j=-padding; j < max_move; j++) {
                for (int k=-padding; k < max_move; k++) {
                    float tmp = 0;
                    for (int l=0; l < output_width; l++) {
                        for (int m=0; m < output_width; m++) {
                            if (not_outside(l*stride+j, m*stride+k, 0, input_width)) {
                                tmp += input[h][l*stride+j][m*stride+k]*output[i][l][m];
                            }
                        }
                    }
                    ker->d_weights[h][i][j][k] += tmp;
                }
            }
        }
    }

    // Input TODO
    if (is_first==1) // Pas besoin de backpropager dans l'input
        return;
    for (int i=0; i < input_depth; i++) {
        for (int j=0; j < input_width; j++) {
            for (int k=0; k < input_width; k++) {
                input[i][j][k] = 0;
            }
        }
    }
    for (int h=0; h < input_depth; h++) {
        for (int i=0; i < output_depth; i++) {
            for (int j=-padding; j < max_move; j++) {
                for (int k=-padding; k < max_move; k++) {
                    for (int l=0; l < output_width; l++) {
                        for (int m=0; m < output_width; m++) {
                            if (not_outside(l*stride+j, m*stride+k, 0, input_width)) {
                                input[h][l*stride+j][m*stride+k] += output[i][l][m]*ker->weights[h][i][j+padding][k+padding];
                            }
                        }
                    }
                }
            }
        }
    }
    for (int i=0; i < input_depth; i++) {
        for (int j=0; j < input_width; j++) {
            for (int k=0; k < input_width; k++) {
                input[i][j][k] = input[i][j][k]*d_function(input_z[i][j][k]);
            }
        }
    }
}

#ifdef __HIPCC__
extern "C"
#endif
void backward_convolution(Kernel_cnn* ker, float*** input, float*** input_z, float*** output, int input_depth, int input_width, int output_depth, int output_width, int activation, int is_first, int kernel_size, int padding, int stride) {
    #ifndef __HIPCC__
    backward_convolution_cpu(ker, input, input_z, output, input_depth, input_width, output_depth, output_width, activation, is_first, kernel_size, padding, stride);
    #else
    backward_convolution_device(ker, input, input_z, output, input_depth, input_width, output_depth, output_width, activation, is_first, kernel_size, padding, stride);
    #endif
}