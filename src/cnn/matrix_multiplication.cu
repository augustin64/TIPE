
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

#ifdef __HIPCC__
/* CUDA memcheck */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#endif

int iDivUp(int hostPtr, int b){
    return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b);
}


float RandFloat(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


void fillMatrixWithRandomValues(float **matrix, int n, int p) {
  for (int i=0; i < n; i++) {
    for (int j=0; j < p; j++) {
        matrix[i][j] = RandFloat(0.0f, 15.0f);
    }
  }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float** create_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(sizeof(float)*p);
    }

    fillMatrixWithRandomValues(matrix, n, p);
    return matrix;
}


float** create_empty_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(p*sizeof(float));
        for (int j=0; j < p; j++) {
            matrix[i][j] = 0.;
        }
    }
    return matrix;
}


#ifdef __HIPCC__
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int n, int p, int q, size_t pitch_m, size_t pitch_n, size_t pitch_p) {
    // 2D Thread ID
    int tx = blockIdx.x*blockDim.x + threadIdx.x;
    int ty = blockIdx.y*blockDim.y + threadIdx.y;
    // Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0.;
    float* M_offset;
    float* N_offset;

    for (int k = 0; k < p; k++) {
        M_offset = (float *)((char*)Md + ty * pitch_m);
        N_offset = (float *)((char*)Nd + k * pitch_n);
    
        Pvalue += M_offset[k] * N_offset[tx];
    }
    // Write the matrix to device memory each thread writes one element
    float* P_offset = (float*)((char*)Pd + ty * pitch_p);
    P_offset[tx] = Pvalue;
}


void matrix_multiplication(float** m1, float** m2, float** result, int n, int p, int q) {
    // Préparation des matrices
    size_t pitch_m1_dev;
    size_t pitch_m2_dev;
    size_t pitch_result_dev;
    float* m1_dev;
    float* m2_dev;
    float* result_dev;
    
    gpuErrchk( hipMallocPitch((void**)&m1_dev, &pitch_m1_dev, p * sizeof(float), n));
    gpuErrchk( hipMemcpy2D(m1_dev, pitch_m1_dev, &m1, p*sizeof(float), p* sizeof(float), n, hipMemcpyHostToDevice));
    
    gpuErrchk( hipMallocPitch((void**)&m2_dev, &pitch_m2_dev, q * sizeof(float), p));
    gpuErrchk( hipMemcpy2D(m2_dev, pitch_m2_dev, &m2, q*sizeof(float), q* sizeof(float), p, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocPitch((void**)&result_dev, &pitch_result_dev, q * sizeof(float), n));

    // Traitement
    dim3 gridSize(iDivUp(n, BLOCKSIZE_x), iDivUp(q, BLOCKSIZE_y));
    dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

    MatrixMulKernel<<<gridSize, blockSize>>>(m1_dev, m2_dev, result_dev, n, p, q, pitch_m1_dev, pitch_m2_dev, pitch_result_dev);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Post-traitement
    for (int i=0; i < q; i++) {
        gpuErrchk( hipMemcpy2D((void*)&(result[i][0]), q*sizeof(float), (const void*)((char*)result_dev + i*pitch_result_dev), pitch_result_dev, sizeof(float)*q, 1, hipMemcpyDeviceToHost));
    }

    gpuErrchk( hipFree(result_dev) );
    gpuErrchk( hipFree(m1_dev) );
    gpuErrchk( hipFree(m2_dev) );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}

#else
void matrix_multiplication(float* m1, float* m2, float* result, int n, int p, int q) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < q; j++) {
            for (int k=0; k < p; k++) {
                result[i*q+j] += m1[i*p+k] + m2[k*q+j];
            }
        }
    }
}
#endif


int main() {
    srand(time(NULL));
    int n = 3;
    int p = 3;
    int q = 3;
    float** matrix1 = create_matrix(n, p);
    float** matrix2 = create_matrix(p, q);
    float** result = create_empty_matrix(n, q);

    clock_t start, end;
    double cpu_time_used;

    start = clock();
    matrix_multiplication(matrix1, matrix2, result, n, p, q);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time used: %lf seconds\n", cpu_time_used);

    print_matrix(matrix1, n, p);
    printf("\n");
    print_matrix(matrix2, p, q);
    printf("\n");
    print_matrix(result, n, q);

    return 0;
}