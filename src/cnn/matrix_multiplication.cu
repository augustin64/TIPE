
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <time.h>

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

#ifdef __HIPCC__
/* CUDA memcheck */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#endif

float random_float(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


void fill_matrix_random(float **matrix, int n, int p) {
  for (int i=0; i < n; i++) {
    for (int j=0; j < p; j++) {
        matrix[i][j] = random_float(0.0f, 15.0f);
    }
  }
}


void print_matrix(float** mat, int n, int p) {
    for (int i=0; i < n; i++) {
        printf("[\t");
        for (int j=0; j < p; j++) {
            printf("%0.1f\t", mat[i][j]);
        }
        printf("]\n");
    }
}


float** create_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(sizeof(float)*p);
    }

    fill_matrix_random(matrix, n, p);
    return matrix;
}


float** create_empty_matrix(int n, int p) {
    float** matrix = (float**)malloc(n*sizeof(float*));
    for (int i=0; i < n; i++) {
        matrix[i] = (float*)malloc(p*sizeof(float));
        for (int j=0; j < p; j++) {
            matrix[i][j] = 0.;
        }
    }
    return matrix;
}


#ifdef __HIPCC__
int i_div_up(int hostPtr, int b){
    return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b);
}


__global__ void matrix_mul_kernel(float* Md, float* Nd, float* Pd, int n, int p, int q, size_t pitch_m, size_t pitch_n, size_t pitch_p) {
    // 2D Thread ID
    int tx = blockIdx.x*blockDim.x + threadIdx.x;
    int ty = blockIdx.y*blockDim.y + threadIdx.y;
    // Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0.;
    float* M_offset;
    float* N_offset;

    for (int k = 0; k < p; k++) {
        M_offset = (float *)((char*)Md + ty * pitch_m);
        N_offset = (float *)((char*)Nd + k * pitch_n);
    
        Pvalue += M_offset[k] * N_offset[tx];
    }
    // Write the matrix to device memory each thread writes one element
    float* P_offset = (float*)((char*)Pd + ty * pitch_p);
    P_offset[tx] = Pvalue;
}


void matrix_multiplication_device(float** m1, float** m2, float** result, int n, int p, int q) {
    // Préparation des matrices
    size_t pitch_m1_dev;
    size_t pitch_m2_dev;
    size_t pitch_result_dev;
    float* m1_dev;
    float* m2_dev;
    float* result_dev;
    
    gpuErrchk( hipMallocPitch((void**)&m1_dev, &pitch_m1_dev, p * sizeof(float), n));
    for (int i=0; i < n; i++) {
        gpuErrchk( hipMemcpy2D((void*)((char*)m1_dev + i*pitch_m1_dev), pitch_m1_dev, (const void*)&(m1[i][0]), p*sizeof(float), p*sizeof(float), 1, hipMemcpyHostToDevice));
    }
    
    gpuErrchk( hipMallocPitch((void**)&m2_dev, &pitch_m2_dev, q * sizeof(float), p));
    for (int i=0; i < p; i++) {
        gpuErrchk( hipMemcpy2D((void*)((char*)m2_dev + i*pitch_m2_dev), pitch_m2_dev, (const void*)&(m2[i][0]), q*sizeof(float), q*sizeof(float), 1, hipMemcpyHostToDevice));
    }

    gpuErrchk( hipMallocPitch((void**)&result_dev, &pitch_result_dev, q * sizeof(float), n));

    // Traitement
    dim3 gridSize(i_div_up(n, BLOCKSIZE_x), i_div_up(q, BLOCKSIZE_y));
    dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

    matrix_mul_kernel<<<gridSize, blockSize>>>(m1_dev, m2_dev, result_dev, n, p, q, pitch_m1_dev, pitch_m2_dev, pitch_result_dev);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Post-traitement
    for (int i=0; i < n; i++) {
        gpuErrchk( hipMemcpy2D((void*)&(result[i][0]), q*sizeof(float), (const void*)((char*)result_dev + i*pitch_result_dev), pitch_result_dev, sizeof(float)*q, 1, hipMemcpyDeviceToHost));
    }

    gpuErrchk( hipFree(result_dev) );
    gpuErrchk( hipFree(m1_dev) );
    gpuErrchk( hipFree(m2_dev) );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif


bool check_cuda_compatibility() {
    #ifdef __HIPCC__
    int nDevices;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&nDevices);
    if (nDevices == 0) {
        printf("Pas d'utilisation du GPU\n\n");
        return false;
    }

    printf("GPUs disponibles:\n");

    for (int i=0; i < nDevices; i++) {
        hipGetDeviceProperties(&prop, i);
        printf(" - %s\n", prop.name);
    }

    hipGetDeviceProperties(&prop, 0);
    printf("Utilisation du GPU: %s\n\n", prop.name);
    return true;
    #else
    printf("Pas d'utilisation du GPU\n\n");
    return false;
    #endif
}


void matrix_multiplication_host(float** m1, float** m2, float** result, int n, int p, int q) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < q; j++) {
            result[i][j] = 0.;
            for (int k=0; k < p; k++) {
                result[i][j] += m1[i][k] + m2[k][j];
            }
        }
    }
}


void matrix_multiplication(float** m1, float** m2, float** result, int n, int p, int q, bool use_cuda) {
    #ifdef __HIPCC__
    if (use_cuda) {
        matrix_multiplication_device(m1, m2, result, n, p, q);
    } else {
        matrix_multiplication_host(m1, m2, result, n, p, q);
    }
    #else
    matrix_multiplication_host(m1, m2, result, n, p, q);
    #endif
}


int main() {
    srand(time(NULL));
    int n = 3;
    int p = 3;
    int q = 3;
    float** matrix1 = create_matrix(n, p);
    float** matrix2 = create_matrix(p, q);
    float** result = create_empty_matrix(n, q);

    clock_t start, end;
    double cpu_time_used;

    start = clock();
    matrix_multiplication(matrix1, matrix2, result, n, p, q, check_cuda_compatibility());
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time used: %lf seconds\n", cpu_time_used);

    print_matrix(matrix1, n, p);
    printf("\n");
    print_matrix(matrix2, p, q);
    printf("\n");
    print_matrix(result, n, q);

    return 0;
}