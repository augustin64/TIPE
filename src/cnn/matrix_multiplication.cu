
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

#ifdef __HIPCC__
/* CUDA memcheck */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#endif


#ifdef __HIPCC__
int i_div_up(int a, int b) { // Partie entière supérieure de a/b
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}


__global__ void matrix_mul_kernel(float* Md, float* Nd, float* Pd, int p, size_t pitch_m, size_t pitch_n, size_t pitch_p) {
    // 2D Thread ID
    int tx = blockIdx.x*blockDim.x + threadIdx.x; // Indice de colonne
    int ty = blockIdx.y*blockDim.y + threadIdx.y; // Indice de ligne
    // Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0.;
    float* M_offset;
    float* N_offset;

    for (int k = 0; k < p; k++) {
        M_offset = (float *)((char*)Md + ty * pitch_m);
        N_offset = (float *)((char*)Nd + k * pitch_n);
    
        Pvalue += M_offset[k] * N_offset[tx];
    }
    // Écrire les résultats des calculs dans la matrice stockée sur le device
    float* P_offset = (float*)((char*)Pd + ty * pitch_p);
    P_offset[tx] = Pvalue;
}


void matrix_multiplication_device(float** m1, float** m2, float** result, int n, int p, int q) {
    // Préparation des matrices
    size_t pitch_m1_dev;
    size_t pitch_m2_dev;
    size_t pitch_result_dev;
    float* m1_dev;
    float* m2_dev;
    float* result_dev;
    
    gpuErrchk( hipMallocPitch((void**)&m1_dev, &pitch_m1_dev, p * sizeof(float), n));
    for (int i=0; i < n; i++) {
        gpuErrchk( hipMemcpy2D((void*)((char*)m1_dev + i*pitch_m1_dev), pitch_m1_dev, (const void*)&(m1[i][0]), p*sizeof(float), p*sizeof(float), 1, hipMemcpyHostToDevice));
    }
    
    gpuErrchk( hipMallocPitch((void**)&m2_dev, &pitch_m2_dev, q * sizeof(float), p));
    for (int i=0; i < p; i++) {
        gpuErrchk( hipMemcpy2D((void*)((char*)m2_dev + i*pitch_m2_dev), pitch_m2_dev, (const void*)&(m2[i][0]), q*sizeof(float), q*sizeof(float), 1, hipMemcpyHostToDevice));
    }

    gpuErrchk( hipMallocPitch((void**)&result_dev, &pitch_result_dev, q * sizeof(float), n));

    // Traitement
    dim3 gridSize(i_div_up(n, BLOCKSIZE_x), i_div_up(q, BLOCKSIZE_y));
    dim3 blockSize(BLOCKSIZE_x, BLOCKSIZE_y);

    matrix_mul_kernel<<<gridSize, blockSize>>>(m1_dev, m2_dev, result_dev, p, pitch_m1_dev, pitch_m2_dev, pitch_result_dev);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Post-traitement
    for (int i=0; i < n; i++) {
        gpuErrchk( hipMemcpy2D((void*)&(result[i][0]), q*sizeof(float), (const void*)((char*)result_dev + i*pitch_result_dev), pitch_result_dev, sizeof(float)*q, 1, hipMemcpyDeviceToHost));
    }

    gpuErrchk( hipFree(result_dev) );
    gpuErrchk( hipFree(m1_dev) );
    gpuErrchk( hipFree(m2_dev) );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
#endif


bool check_cuda_compatibility() {
    #ifdef __HIPCC__
    int nDevices;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&nDevices);
    if (nDevices == 0) {
        printf("Pas d'utilisation du GPU\n\n");
        return false;
    }

    printf("GPUs disponibles:\n");

    for (int i=0; i < nDevices; i++) {
        hipGetDeviceProperties(&prop, i);
        printf(" - %s\n", prop.name);
    }

    hipGetDeviceProperties(&prop, 0);
    printf("Utilisation du GPU: %s\n\n", prop.name);
    return true;
    #else
    printf("Pas d'utilisation du GPU\n\n");
    return false;
    #endif
}


void matrix_multiplication_host(float** m1, float** m2, float** result, int n, int p, int q) {
    for (int i=0; i < n; i++) {
        for (int j=0; j < q; j++) {
            result[i][j] = 0.;
            for (int k=0; k < p; k++) {
                result[i][j] += m1[i][k] * m2[k][j];
            }
        }
    }
}


void matrix_multiplication(float** m1, float** m2, float** result, int n, int p, int q, bool use_cuda) {
    #ifdef __HIPCC__
    if (use_cuda) {
        matrix_multiplication_device(m1, m2, result, n, p, q);
    } else {
        matrix_multiplication_host(m1, m2, result, n, p, q);
    }
    #else
    matrix_multiplication_host(m1, m2, result, n, p, q);
    #endif
}