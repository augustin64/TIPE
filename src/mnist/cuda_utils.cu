#include <stdio.h>
#include <stdlib.h>


int*** copy_images_cuda(int*** images, int nb_images, int width, int height) {
    int*** images_cuda;
    hipMalloc((int****)&images_cuda, sizeof(int**)*nb_images);
    hipMemcpy((int****)&images_cuda, sizeof(int**)*nb_images, images);

    for (int i=0; i < nb_images; i++) {
        hipMalloc((int***)&images_cuda[i], sizeof(int**)*nb_images);
        hipMemcpy((int***)&images_cuda[i], sizeof(int**)*nb_images, images[i]);
            for (int j=0; j < height; j++) {
                hipMalloc((int**)&images_cuda[i][j], sizeof(int*)*width);
                hipMemcpy((int**)&images_cuda[i][j], sizeof(int*)*width, images[i][j]);
            }
    }
    return images_cuda;
}




unsigned int* copy_labels_cuda(unsigned int* labels) {
    unsigned int* labels_cuda;
    hipMalloc((unsigned int**)&labels_cuda, sizeof(labels));
    hipMemcpy((unsigned int**)&labels_cuda, sizeof(labels), labels);
    return labels_cuda;
}