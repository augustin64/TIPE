#include <stdio.h>
#include <stdlib.h>

#include "include/cuda_utils.h"

int*** copy_images_cuda(int*** images, int nb_images, int width, int height) {
    int*** images_cuda;
    hipMalloc(&images_cuda, (size_t)sizeof(int**)*nb_images);
    hipMemcpy(images_cuda, &images, (size_t)sizeof(int**)*nb_images, hipMemcpyHostToDevice);

    for (int i=0; i < nb_images; i++) {
        hipMalloc(&images_cuda[i], sizeof(int**)*nb_images);
        hipMemcpy(images_cuda[i], &images[i], sizeof(int**)*nb_images, hipMemcpyHostToDevice);
            for (int j=0; j < height; j++) {
                hipMalloc((int**)&images_cuda[i][j], sizeof(int*)*width);
                hipMemcpy(images_cuda[i][j], &images[i][j], sizeof(int*)*width, hipMemcpyHostToDevice);
            }
    }
    return images_cuda;
}


unsigned int* copy_labels_cuda(unsigned int* labels) {
    unsigned int* labels_cuda;
    hipMalloc(&labels_cuda, (size_t)sizeof(labels));
    hipMemcpy(labels_cuda, &labels, sizeof(labels), hipMemcpyHostToDevice);
    return labels_cuda;
}


void check_cuda_compatibility() {
    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        printf("%s\n", hipGetErrorString(err));
        exit(1);
    } else {
        printf("CUDA-capable device is detected\n");
    }
}